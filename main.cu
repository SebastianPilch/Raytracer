#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parallel_functions.cuh"
#include "Material.cuh"
#include "SaveAsBMP.cuh"
#include<cmath>


int main() {

    ///////////////////////////////////////////////
    //
    //wczytanie obiektów .obj
    //
    //////////////////////////////////////////////////

    int object_couter = 0;
    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, object_couter, "../../../helpers/Studia_z_budownictwa_budujemy_mosty_2.obj");


    ///////////////////////////////////////////////
    //
    //wybór obiektu
    //
    //////////////////////////////////////////////////
    int Vert_NUM = vert_num3;
    int Face_NUM = face_num3;
    int Normal_NUM = normal_num3;


    cout << object_couter << endl;

    Pointer_storage liczony_objekt = pociety_walec;
    ///////////////////////////////////////////////
    //
    //przepisanie wskaźników
    //
    //////////////////////////////////////////////////
    float** Planes = new float* [Face_NUM];
    Planes[0] = new float[Face_NUM * 4];

    float** Verticies = new float* [Vert_NUM];
    Verticies[0] = new float[Vert_NUM * 3];

    float** Normals = new float* [Normal_NUM];
    Normals[0] = new float[Normal_NUM * 3];



    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;
    int* Object_to_Face = liczony_objekt.Object_to_Face;
    int* Object_to_Vertex = liczony_objekt.Object_to_Vertex;

    float* Distances = new float[WIDTH * HEIGHT * Face_NUM];
    float* Colors = new float[WIDTH * HEIGHT * 3];
    float* shadows = new float[WIDTH * HEIGHT * 3];

    int* start_face_at_index = new int[Face_NUM];
    float* Intersections = new float[WIDTH * HEIGHT * 3];
    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) {
        Length_to_Allocate_Faces += number_of_vertices_in_one_face[i];
    }

    int** Faces = new int* [Face_NUM];
    Faces[0] = new int[Length_to_Allocate_Faces];

    int current_index = 0;
    for (int i = 1; i < Face_NUM + 1; i++) {
        Faces[i] = Faces[0] + current_index + number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i - 1] = current_index;
        current_index += number_of_vertices_in_one_face[i - 1];
        Planes[i] = Planes[0] + i * 4;


        //cout << i-1 << "  :  " << start_face_at_index[i-1] << endl;
    }

    for (int i = 1; i < Vert_NUM; i++) {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++) {
        Normals[i] = Normals[0] + i * 3;
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < 4; j++)
        {
            Planes[i][j] = liczony_objekt.Planes[i][j];
        }
    }
    for (int i = 0; i < Vert_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Verticies[i][j] = liczony_objekt.Vertices[i][j];
    }
    for (int i = 0; i < Normal_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Normals[i][j] = liczony_objekt.Normals[i][j];
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < liczony_objekt.Face_size[i]; j++) {
            Faces[i][j] = liczony_objekt.Faces[i][j];
        }
    }

    ///////////////////////////////////////////////
    //
    //utworzenie materiałów
    //
    //////////////////////////////////////////////////
    Material* Materials = new Material[object_couter];
    //trawa
    Materials[5] = Material(0.6f, 0.8f, 0.3f,   // diffuse (bright green)
        0.7f, 0.9f, 0.4f,   // specular (light green)
        0.3f, 0.4f, 0.2f,   // ambient (dark green)
        1.0f,               // alpha
        4.0f,              // shininess  
        0.0f);             // reflectivity

    //woda

    Materials[2] = Material(0.3f, 0.6f, 0.8f,   // diffuse (light blue)
        0.5f, 0.7f, 0.9f,   // specular (light blue)
        0.2f, 0.3f, 0.4f,   // ambient (dark blue)
        1.0f,               // alpha
        4.0f,              // shininess  
        0.3f);             // reflectivity

    //pnie

    Materials[4] = Material(0.5f, 0.3f, 0.1f,   // diffuse (brown)
        0.6f, 0.4f, 0.2f,   // specular (light brown)
        0.3f, 0.2f, 0.1f,   // ambient (dark brown)
        1.0f,               // alpha
        4.0f,              // shininess  
        0.0f);             // reflectivity

    //drzewa

    Materials[3] = Material(0.1f, 0.4f, 0.1f,   // diffuse (dark green)
        0.2f, 0.5f, 0.2f,   // specular (light green)
        0.05f, 0.2f, 0.05f, // ambient (very dark green)
        1.0f,               // alpha
        4.0f,              // shininess  
        0.0f);             // reflectivity

    //most

    Materials[1] = Material(0.6f, 0.6f, 0.6f,   // diffuse (metallic grey)
        0.8f, 0.8f, 0.8f,   // specular (bright grey)
        0.3f, 0.3f, 0.3f,   // ambient (dark grey)
        1.0f,               // alpha
        4.0f,              // shininess  
        0.15f);             // reflectivity
    //skała
    Materials[0] = Material(0.8f, 0.8f, 0.8f,   // diffuse (metallic grey)
        0.8f, 0.8f, 0.8f,   // specular (bright grey)
        0.3f, 0.3f, 0.3f,   // ambient (dark grey)
        1.0f,               // alpha
        1.0f,              // shininess  
        0.0f);             // reflectivity
    ///////////////////////////////////////////////
    //
    //Alokacja i przekazanie danych do karty
    //
    //////////////////////////////////////////////////

    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    int* d_Object_to_Vertex;
    int* d_Object_to_Face;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;
    float* d_closest_interesections;
    float* d_shadows;
    Material* d_Materials;

    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    hipMalloc(&d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float));
    hipMalloc(&d_closest_interesections, WIDTH * HEIGHT * 3 * sizeof(float));
    hipMalloc(&d_Object_to_Vertex, Vert_NUM * sizeof(int));
    hipMalloc(&d_Object_to_Face, Face_NUM * sizeof(int));
    hipMalloc(&d_shadows, WIDTH* HEIGHT * 3 * sizeof(float));


    hipMemcpy(d_Faces, Faces[0], Length_to_Allocate_Faces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Planes, Planes[0], 4 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Normals, Normals[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Vertices, Verticies[0], 3 * Vert_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_vertices_in_one_face, number_of_vertices_in_one_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_normal_index_to_face, normal_index_to_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_face_at_index, start_face_at_index, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Object_to_Vertex, Object_to_Vertex, Vert_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Object_to_Face, Object_to_Face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);

    ///////////////////////////////////////////////
    //
    //Tranformacja obiektu rotacja/skala/przesunięcie
    //
    //////////////////////////////////////////////////

    int threadsPerBlock = 256;
    int blocksPerGrid = (Vert_NUM + threadsPerBlock - 1) / threadsPerBlock;
    float TranslateX = 13.0f;
    float TranslateY = 0.0f;
    float TranslateZ = -6.0f;
    float rotateX = 0.0f;
    float rotateY = 135.0f;
    float rotateZ = 0.0f;
    float scaleX = 1.5f;
    float scaleY = -1.5f;
    float scaleZ = -1.5f;

    int index = 0;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 1;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 2;
    Transform <<< blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 3;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 4;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 5;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    threadsPerBlock = 256;
    blocksPerGrid = (Face_NUM + threadsPerBlock - 1) / threadsPerBlock;
    Update_normals_and_Planes << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, d_Faces, d_Object_to_Face, index, d_Normals, d_Planes, d_number_of_vertices_in_one_face, d_normal_index_to_face, d_start_face_at_index, Face_NUM, Normal_NUM);
    hipDeviceSynchronize();

    ///////////////////////////////////////////////
    //
    //  Wyznaczanie Promieni, uderzenia i dystanse
    //
    //////////////////////////////////////////////////
    int reflecions = 0;
    double focal_length = 10;
    point3 h_camera_center(120.0, -70.0, -70.0);
    point3  h_camera_focal(120.0 / 2, -70.0 / 2, -70.0 / 2);
    vec3 world_light_dir = vec3(-0.7,1,-1);
    point3* d_camera_center;
    point3* d_camera_focal;
    vec3* d_world_light_dir;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMalloc((void**)&d_world_light_dir, sizeof(vec3));

    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_world_light_dir, &world_light_dir, sizeof(point3), hipMemcpyHostToDevice);



    dim3 blockDim(8, 8, 8);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    ///////////////////////////////////////////////
    //
    // wyznaczenie promieni
    //
    ///////////////////////////////////////////////
    hipMemcpy(d_ray, h_ray[0], WIDTH * HEIGHT * sizeof(ray), hipMemcpyHostToDevice);


    Generate_rays <<< gridDim, blockDim >>> (d_ray, focal_length, d_camera_center, d_camera_focal);
    hipDeviceSynchronize();

    ///////////////////////////////////////////////
    //
    // wyznaczenie wektora dystansów
    //
    ///////////////////////////////////////////////

    Generate_distances << <gridDim, blockDim >> > (d_ray, d_camera_center, d_closest_interesections, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, reflecions);
    hipDeviceSynchronize();
    reflecions += 1;




    ///////////////////////////////////////////////
    //
    // dobór najbliższej ściany z wektora dystansów
    //
    ////////////////////////////////////////////////


    int* d_close_indexes;
    float* d_colors;

    hipMalloc(&d_Materials, object_couter * sizeof(Material));
    hipMalloc(&d_close_indexes, WIDTH * HEIGHT * sizeof(int));
    hipMalloc(&d_colors, WIDTH * HEIGHT * 3 * sizeof(float));


    hipMemcpy(d_Materials, Materials, object_couter * sizeof(Material), hipMemcpyHostToDevice);

    Choose_closest <<< gridDim, blockDim >>> (d_distances, Face_NUM, d_colors, d_Planes, d_Materials, d_ray, d_Object_to_Face, d_close_indexes, d_closest_interesections, d_world_light_dir);
    hipDeviceSynchronize();

    int* Reflected_surface = new int[WIDTH * HEIGHT];

    hipMemcpy(Colors, d_colors, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Reflected_surface, d_close_indexes, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

    saveAsBMP(Colors, WIDTH, HEIGHT, "normals_image.bmp");
    ///////////////////////////////////////////////
    //
    // Dodanie cienia
    //
    ////////////////////////////////////////////////


    Add_shadows <<< gridDim, blockDim >> > (d_closest_interesections, d_shadows, d_normal_index_to_face, d_number_of_vertices_in_one_face, d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_world_light_dir);
    hipDeviceSynchronize();

    hipMemcpy(shadows, d_shadows, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);


    saveAsBMP(shadows, WIDTH, HEIGHT, "shadowes_image.bmp");




    ///////////////////////////////////////////////
    //
    // druga iteracja promieni
    //
    ////////////////////////////////////////////////
    Update_rays << < gridDim, blockDim >>> (d_ray, d_closest_interesections, d_close_indexes, d_normal_index_to_face, d_Normals);
    hipDeviceSynchronize();


    Generate_distances << < gridDim, blockDim >> > (d_ray, d_camera_center, d_closest_interesections, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, reflecions);
    hipDeviceSynchronize();
    reflecions += 1;


    hipMemcpy(d_Materials, Materials, object_couter * sizeof(Material), hipMemcpyHostToDevice);

    Choose_closest << <gridDim, blockDim >> > (d_distances, Face_NUM, d_colors, d_Planes, d_Materials, d_ray, d_Object_to_Face, d_close_indexes, d_closest_interesections, d_world_light_dir);
    hipDeviceSynchronize();


    float* Reflections = new float[WIDTH * HEIGHT * 3];

    hipMemcpy(Reflections, d_colors, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);
    saveAsBMP(Reflections, WIDTH, HEIGHT, "reflected_image.bmp");


    cout << "skłądanie obrazów" << endl;

    for (int i = 0; i < WIDTH * HEIGHT; i++)
    {
        if (Reflected_surface[i] != -1) 
        {
            Material Mat;
            Mat = Materials[Object_to_Face[Reflected_surface[i]]];
            Reflections[i * 3] = Mat.Reflectivity * Reflections[i * 3] + Colors[i * 3] * (1 - Mat.Reflectivity);
            Reflections[i * 3 + 1] = Mat.Reflectivity * Reflections[i * 3 + 1] + Colors[i * 3 + 1] * (1 - Mat.Reflectivity);
            Reflections[i * 3 + 2] = Mat.Reflectivity * Reflections[i * 3 + 2] + Colors[i * 3 + 2] * (1 - Mat.Reflectivity);
            if (shadows[i * 3] >= 1) { Reflections[i * 3] -= 0.1f; if (Reflections[i * 3] < 0) { Reflections[i * 3] = 0.0f; } }
            if (shadows[i * 3 + 1] >= 1) { Reflections[i * 3 + 1] -= 0.1f; if (Reflections[i * 3 + 1] < 0) { Reflections[i * 3 + 1] = 0.0f; } }
            if (shadows[i * 3 + 2] >= 1) { Reflections[i * 3 + 2] -= 0.1f; if (Reflections[i * 3 + 2] < 0) { Reflections[i * 3 + 2] = 0.0f; } }
        
        }

    }
    saveAsBMP(Reflections, WIDTH, HEIGHT, "Complete_image.bmp");


    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    hipFree(d_closest_interesections);

    free(h_ray[0]);
    free(h_ray);
    delete[] Colors;

    return 0;
}