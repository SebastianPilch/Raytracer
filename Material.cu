#include "hip/hip_runtime.h"
#include "material.cuh"

__host__ __device__ Material::Material() {
    Diffuse[0] = 0.0f; Diffuse[1] = 0.0f; Diffuse[2] = 0.0f;
    Specular[0] = 0.0f; Specular[1] = 0.0f; Specular[2] = 0.0f;
    Ambient[0] = 0.0f; Ambient[1] = 0.0f; Ambient[2] = 0.0f;
    Alpha = 1.0f;
    Shininess = 0.0f;
}

__host__ __device__ Material::Material(const float diffuse[3], const float specular[3], const float ambient[3], float alpha, float shininess) {
    for (int i = 0; i < 3; i++) {
        Diffuse[i] = clamp(diffuse[i], 0.0f, 1.0f);
        Specular[i] = clamp(specular[i], 0.0f, 1.0f);
        Ambient[i] = clamp(ambient[i], 0.0f, 1.0f);
    }
    Alpha = clamp(alpha, 0.0f, 1.0f);
    Shininess = clamp(shininess, 0.0f, 1.0f);
}

__host__ __device__ float Material::clamp(float value, float min, float max) {
    if (value < min) return min;
    if (value > max) return max;
    return value;
}
