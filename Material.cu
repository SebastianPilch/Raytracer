#include "hip/hip_runtime.h"
#include "material.cuh"

__host__ __device__ Material::Material() {
    Diffuse[0] = 0.0f; Diffuse[1] = 0.0f; Diffuse[2] = 0.0f;
    Specular[0] = 0.0f; Specular[1] = 0.0f; Specular[2] = 0.0f;
    Ambient[0] = 0.0f; Ambient[1] = 0.0f; Ambient[2] = 0.0f;
    Alpha = 1.0f;
    Shininess = 0.0f;
}

__host__ __device__ Material::Material(float d0, float d1, float d2,
                                       float s0, float s1, float s2,
                                       float a0, float a1, float a2,
                                       float alpha, float shininess) {
    Diffuse[0] = clamp(d0, 0.0f, 1.0f);
    Diffuse[1] = clamp(d1, 0.0f, 1.0f);
    Diffuse[2] = clamp(d2, 0.0f, 1.0f);

    Specular[0] = clamp(s0, 0.0f, 1.0f);
    Specular[1] = clamp(s1, 0.0f, 1.0f);
    Specular[2] = clamp(s2, 0.0f, 1.0f);

    Ambient[0] = clamp(a0, 0.0f, 1.0f);
    Ambient[1] = clamp(a1, 0.0f, 1.0f);
    Ambient[2] = clamp(a2, 0.0f, 1.0f);

    Alpha = clamp(alpha, 0.0f, 1.0f);
    Shininess = clamp(shininess, 0.0f, 1.0f);
}

__host__ __device__ float Material::clamp(float value, float min, float max) {
    if (value < min) return min;
    if (value > max) return max;
    return value;
}
