#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include "ImportObj.cuh"
//#include <hip/hip_runtime.h>
//#include <>

using namespace std;

size_t split(const std::string& txt, std::vector<std::string>& strs, char ch)
{
    size_t pos = txt.find(ch);
    size_t initialPos = 0;
    strs.clear();

    // Decompose statement
    while (pos != std::string::npos) {
        strs.push_back(txt.substr(initialPos, pos - initialPos));
        initialPos = pos + 1;

        pos = txt.find(ch, initialPos);
    }

    // Add the last one
    strs.push_back(txt.substr(initialPos, std::min(pos, txt.size()) - initialPos + 1));

    return strs.size();
}
Pointer_storage GetDataFromObj(int& Vertices_coords_size, int& Face_numer, int& Normals_size, int& object_counter, string file_path)
{
    ifstream plik(file_path);
    if (!plik.is_open())
    {
        cerr << "Nie uda³o siê otworzyæ pliku: " << file_path << endl;
        return Pointer_storage();
    }

    string linia;
    vector<string> Splited_line;
    vector<string> Backshlash_face_split;
    size_t newSize = 1;
    float** vertices = new float* [Vertices_coords_size];
    for (size_t i = 0; i < Vertices_coords_size; ++i) {
        vertices[i] = new float[3];
        for (size_t j = 0; j < 3; ++j) { vertices[i][j] = 0; }
    }
    float** faces_normals = new float* [Normals_size];
    for (size_t i = 0; i < Normals_size; ++i) {
        faces_normals[i] = new float[3];
    }
    float** Planes_to_faces = new float* [Face_numer];
    for (size_t i = 0; i < Face_numer; ++i) {
        Planes_to_faces[i] = new float[3];
    }
    int** vertices_to_faces = new int* [Face_numer];
    for (size_t i = 0; i < Face_numer; ++i) {
        vertices_to_faces[i] = new int[3];
    }

    int* noramls_index_to_face = new int[Face_numer];
    int* vertices_in_one_face = new int[Face_numer];
    int* Object_idx_to_vertex = new int[Vertices_coords_size];
    int* Object_idx_to_face = new int[Face_numer];





    float** newVertices = nullptr;
    float** newNormals = nullptr;
    float** newFaces = nullptr;
    float** newPlanes = nullptr;

    int* new_face_lengths = nullptr;
    int* new_normals_index_to_face = nullptr;
    int* material_to_face = nullptr;


    int* new_Object_idx_to_vertex = nullptr;
    int* new_Object_idx_to_face = nullptr;



    int index_f = 0;
    int index_n = 0;
    int len_v = 0;

    size_t newSize_V;
    size_t newSize_F;



    while (getline(plik, linia))
    {
        split(linia, Splited_line, ' ');

        if (Splited_line[0] == "o")
        {
            object_counter++;
        }
        if (Splited_line[0] == "v") {
            if (len_v >= Vertices_coords_size) {
                newSize_V = Vertices_coords_size * 2;
                float** newVertices = new float* [newSize_V];
                int* new_Object_idx_to_vertex = new int[newSize_V];
                for (size_t i = 0; i < newSize_V; ++i) {
                    newVertices[i] = new float[3];
                }
                for (size_t i = 0; i < len_v; ++i) {
                    new_Object_idx_to_vertex[i] = Object_idx_to_vertex[i];
                    for (size_t j = 0; j < 3; ++j) {
                        newVertices[i][j] = vertices[i][j];
                    }
                    delete[] vertices[i];

                }
                delete[] vertices;
                delete[] Object_idx_to_vertex;
                vertices = newVertices;
                Object_idx_to_vertex = new_Object_idx_to_vertex;
                for (size_t i = 0; i < newSize_V; i++) {
                    vertices[i] = newVertices[i];
                }
                Vertices_coords_size = newSize_V;
            }
            vertices[len_v] = new float[3];
            vertices[len_v][0] = (float)stof(Splited_line[1]);
            vertices[len_v][1] = (float)stof(Splited_line[2]);
            vertices[len_v][2] = (float)stof(Splited_line[3]);

            Object_idx_to_vertex[len_v] = object_counter;

            len_v++;
        }


        //cout << "v - posz³o" << endl;

        if (Splited_line[0] == "vn")
        {
            if (index_n >= Normals_size)
            {
                size_t newSize = Normals_size * 2;
                float** newNormals = new float* [newSize];
                for (size_t i = 0; i < index_n; ++i) {
                    newNormals[i] = new float[3];
                    for (size_t j = 0; j < 3; ++j) {
                        newNormals[i][j] = faces_normals[i][j];
                    }
                    delete[] faces_normals[i];
                }
                delete[] faces_normals;
                faces_normals = newNormals;
                for (size_t i = 0; i < newSize; ++i) {
                    faces_normals[i] = newNormals[i];
                }
                Normals_size = newSize;

            }
            faces_normals[index_n] = new float[3];
            faces_normals[index_n][0] = stof(Splited_line[1]);
            faces_normals[index_n][1] = stof(Splited_line[2]);
            faces_normals[index_n][2] = stof(Splited_line[3]);
            index_n++;
        }




        if (Splited_line[0] == "f")
        {
            if (index_f >= Face_numer) {
                newSize_F = Face_numer * 2;
                int* new_face_lengths = new int[newSize_F];
                int* new_Object_idx_to_face = new int[newSize_F];
                int* new_normals_index_to_face = new int[newSize_F];
                float** NewPlanes = new float* [newSize_F];
                int** newFaces = new int* [newSize_F];

                for (size_t i = 0; i < index_f; i++) {
                    NewPlanes[i] = new float[4];
                    newFaces[i] = new int[vertices_in_one_face[i]];
                    new_face_lengths[i] = vertices_in_one_face[i];
                    new_Object_idx_to_face[i] = Object_idx_to_face[i];
                    new_normals_index_to_face[i] = noramls_index_to_face[i];
                    for (size_t j = 0; j < 4; ++j) {
                        NewPlanes[i][j] = Planes_to_faces[i][j];
                    }
                    delete[] Planes_to_faces[i];

                    for (size_t j = 0; j < vertices_in_one_face[i]; ++j) {
                        newFaces[i][j] = vertices_to_faces[i][j];
                    }
                    delete[] vertices_to_faces[i];
                }


                delete[] vertices_in_one_face;
                delete[] Object_idx_to_face;
                vertices_in_one_face = new_face_lengths;
                Object_idx_to_face = new_Object_idx_to_face;
                delete[] noramls_index_to_face;
                noramls_index_to_face = new_normals_index_to_face;
                delete[] Planes_to_faces;
                Planes_to_faces = NewPlanes;
                for (size_t i = 0; i < newSize_F; i++) {
                    Planes_to_faces[i] = NewPlanes[i];
                }

                delete[] vertices_to_faces;
                vertices_to_faces = newFaces;
                for (size_t i = 0; i < newSize_F; i++) {
                    vertices_to_faces[i] = newFaces[i];
                }
                Face_numer = newSize_F;
            }

            vertices_in_one_face[index_f] = (int)(Splited_line.size() - 1);
            vertices_to_faces[index_f] = new int[vertices_in_one_face[index_f]];
            for (int i = 0; i < vertices_in_one_face[index_f]; i++)
            {
                vertices_to_faces[index_f][i] = (int)stof(Splited_line[i + 1]);
            }
            split(Splited_line[1], Backshlash_face_split, '/');
            noramls_index_to_face[index_f] = (int)stof(Backshlash_face_split[2]);

            int current_face_normal = noramls_index_to_face[index_f];
            float x = vertices[vertices_to_faces[index_f][0]][0];
            float y = vertices[vertices_to_faces[index_f][0]][1];
            float z = vertices[vertices_to_faces[index_f][0]][2];
            float nor_x = faces_normals[current_face_normal - 1][0];
            float nor_y = faces_normals[current_face_normal - 1][1];
            float nor_z = faces_normals[current_face_normal - 1][2];
            Planes_to_faces[index_f] = new float[4];
            Planes_to_faces[index_f][0] = nor_x;
            Planes_to_faces[index_f][1] = nor_y;
            Planes_to_faces[index_f][2] = nor_z;
            Planes_to_faces[index_f][3] = -(nor_x * x + nor_y * y + nor_z * z);

            Object_idx_to_face[index_f] = object_counter;

            //cout << nor_x  <<  " " << x << " " << nor_y <<  " " << y <<  " " << nor_z << " " << z << endl;

            index_f++;
        }
    }

    Vertices_coords_size = len_v;
    Face_numer = index_f;
    Normals_size = index_n;


    plik.close();

    delete[] newVertices;
    delete[] newNormals;
    delete[] newFaces;
    delete[] newPlanes;
    delete[] new_face_lengths;
    delete[] new_normals_index_to_face;

    return Pointer_storage(vertices, faces_normals, vertices_to_faces, Planes_to_faces, vertices_in_one_face, noramls_index_to_face);
}


void Print_Import_data(Pointer_storage object, int ver_size, int nor_size, int Face_size)
{
    for (int i = 0; i < ver_size; i++)
    {
        cout << "Vertex " << i + 1 << " :  " << object.Vertices[i][0] << " ,  " << object.Vertices[i][1] << " ,  "
            << object.Vertices[i][2] << endl;
    }

    cout << endl << endl;


    for (int i = 0; i < nor_size; i++)
    {
        cout << "Normal " << i + 1 << " :  " << object.Normals[i][0] << " ,  " << object.Normals[i][1] << " ,  "
            << object.Normals[i][2] << endl;
    }
    cout << endl << endl;


    for (int i = 0; i < Face_size; i++)
    {
        cout << "Vertices creating face " << i + 1 << " :  " << object.Face_size[i] << endl;
    }

    cout << endl << endl;

    for (int i = 0; i < Face_size; i++)
    {
        cout << "Normal index to face " << i + 1 << " :  " << object.Face_to_Normal[i] << endl;
    }
    cout << endl << endl;


    for (int i = 0; i < Face_size; i++)
    {
        cout << "Plane " << i + 1 << " :  ";
        for (int j = 0; j < 4; j++)
        {
            cout << object.Planes[i][j] << " ,  ";
        }
        cout << endl;
    }


    cout << endl << endl;

    for (int i = 0; i < Face_size; i++)
    {
        cout << "Face " << i + 1 << " :  ";
        for (int j = 0; j < object.Face_size[i]; j++)
        {
            cout << object.Faces[i][j] << " ,  ";
        }
        cout << endl;
    }

}