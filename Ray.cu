#include "hip/hip_runtime.h"
#include <map>
#include <vector>
#include "vec3.cuh"
#include "Ray.cuh"



using namespace std;
double positiveInfinity = INFINITY;
double negativeInfinity = -INFINITY;
__host__ __device__ ray::ray() {};
__host__ __device__ ray::ray(const point3& origin, const vec3& direction) : orig(origin), dir(direction) {};
__host__ __device__ point3 ray:: at(double t) const { return this->dir*t + this->orig; }
__host__ __device__ point3 ray::findIntersection(const Plane& plane) const
{
double t = -(plane.A * this->orig.x() + plane.B * this->orig.y() + plane.C * this->orig.z() + plane.D) /
    (plane.A * this->dir.x() + plane.B * this->dir.y() + plane.C * this->dir.z());
    point3 intersection(at(t));
    return intersection;
}

__host__ __device__ std::ostream& operator<<(std::ostream& out, const ray& r) {
    return out <<'[' << r.dir <<"]*t + [" << r.orig <<"]";
}


//bool Face_hit(float* pl, const ray& r, int polygon_langht ,int* polygon, float** vertices_coords)
//{
//    Plane plane = Plane((double)pl[0], (double)pl[1], (double)pl[2], (double)pl[3]);
//    point3 intersection = r.findIntersection(plane);
//    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
//        intersection[1] == -INFINITY == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY) {
//        return false;
//    }
//    vec3 edge;
//    for (size_t i = start_index_per_face; i < polygon_langht; ++i) {
//        size_t next_index = (i + 1) % polygon_langht;
//        point3 current_vertex = point3((double)vertices_coords[polygon[i] - 1][0], (double)vertices_coords[polygon[i] - 1][1], (double)vertices_coords[polygon[i] - 1][2]);
//        point3 next_vertex = point3((double)vertices_coords[polygon[next_index] - 1][0], (double)vertices_coords[polygon[next_index] - 1][1], (double)vertices_coords[polygon[next_index] - 1][2]);
//        edge = next_vertex - current_vertex;
//        vec3 vp = intersection - current_vertex;
//        vec3 n = crossProduct_(edge, vp);
//        vec3 normal = vec3(plane.A, plane.B, plane.C);
//        if (dotProduct_(n, normal) < 0) {
//            return false;
//        }
//    }
//
//
//};

__global__ void Face_hit(float* pl, const ray& r, int* polygon_langht, int* polygon, float* vertices_coords, int Face_number, int* start_index_per_face) {

    extern __shared__ float distances[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    for (int i = tid; i < Face_number; i += blockDim.x) {
        distances[i] = 0.0f;
    }
    __syncthreads();


    if (idx >= Face_number) return;

    Plane plane = Plane((double)pl[0], (double)pl[1], (double)pl[2], (double)pl[3]);
    point3 intersection = r.findIntersection(plane);
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY) {
    }

    vec3 edge;
    for (size_t i = start_index_per_face[idx]; i < start_index_per_face[idx] + polygon_langht[idx]; ++i) {


        size_t next_index;
        if (i + 1 > start_index_per_face[idx] + polygon_langht[idx])
        {
            next_index = i + 1;

        }
        else
        {
            next_index = start_index_per_face[idx];

        }

        int vertex_index = polygon[i] - 1;
        int vertex_next = polygon[next_index] - 1;

        point3 next_vertex = point3(
            (double)vertices_coords[3 * vertex_index],
            (double)vertices_coords[3 * vertex_index + 1],
            (double)vertices_coords[3 * vertex_index + 2]
        );

        point3 current_vertex = point3(
            (double)vertices_coords[3 * vertex_next],
            (double)vertices_coords[3 * vertex_next + 1],
            (double)vertices_coords[3 * vertex_next + 2]
        );


        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) {
            return;
        }
    }

}