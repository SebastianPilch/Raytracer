#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"
#include <cmath>

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

__global__ void Generate_rays(ray* viewport_rays, double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index,
    int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances, float* d_closest_angles) {



    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;

    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) return;


    float aspect_ratio = float(WIDTH) / float(HEIGHT);
    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_dist = focal_length;

    vec3 w = (*camera_center - *camera_focal) / ((*camera_center - *camera_focal).length());
    vec3 u = (crossProduct_(vec3(0, 1, 0), w)) / ((crossProduct_(vec3(0, 1, 0), w)).length());
    vec3 v = crossProduct_(w,u);

    vec3 horizontal = u * viewport_width;
    vec3 vertical = v * viewport_height;
    vec3 lower_left_corner = *camera_center - horizontal / 2 - vertical / 2 - w * focal_dist;

    float u_offset = float(i) / (WIDTH - 1);
    float v_offset = float(j) / (HEIGHT - 1);
    vec3 pixel_center = lower_left_corner + horizontal * u_offset + vertical * v_offset;

/*    vec3 viewport_upper_left = *camera_center - vec3(focal_length, focal_length, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V) + *camera_focal;*/


    vec3 ray_direction = pixel_center - *camera_center;
    ray UV_ray = ray(*camera_center, ray_direction);
    viewport_rays[j * HEIGHT + i] = ray(*camera_center, ray_direction);

    __syncthreads();

    Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
    point3 intersection = UV_ray.findIntersection(plane);
    bool Is_Hitten_correct = true;
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY)
    {
        Is_Hitten_correct = false;
    }

    vec3 edge;
    size_t next_index;
    for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
    {
        size_t next_index;
        if (idx + 1 >= start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = start_face_at_index[f];
        }
        else
        {
            next_index = idx + 1;
        }
        int vertex_index = d_Faces[idx] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );

        __syncthreads();
        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) 
        {
            Is_Hitten_correct = false;
        }
        if (current_vertex[0] == 0 && current_vertex[1] == 0 && current_vertex[2] == 0)
        {
            //printf("%d", vertex_index);
        }
    }
    vec3 dis = *camera_center - intersection;

    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
        //printf("%f  %f  %f \n", intersection[0], intersection[1], intersection[2]);

    }
    else
    {
        float distance = sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = distance;

    }


    __syncthreads();
}

__global__ void Choose_closest(float* d_distances,int Face_NUM, float* d_closest_normals, float* d_Planes, Material* Mats, ray* rays)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < WIDTH && j < HEIGHT) {
        int closest = -1;
        float lowest_distance = INFINITY;

        for (int f = 0; f < Face_NUM; f++)
        {
            if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] < lowest_distance)
            {
                closest = f;
                lowest_distance = d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f];
            }
        }

        Plane plane = Plane((double)d_Planes[closest * 4], (double)d_Planes[closest * 4 + 1], (double)d_Planes[closest * 4 + 2], (double)d_Planes[closest * 4 + 3]);

        if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] == INFINITY || d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] < 0.0f)
        {
            d_closest_normals[(j * WIDTH + i) * 3 + 0] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = 0.0f;
        }
        else
        {
            vec3 normal = vec3((float)plane.A, (float)plane.B, (float)plane.C);
            float norm_length = sqrt(normal.x() * normal.x() + normal.y() * normal.y() + normal.z() * normal.z());
            normal = normal / norm_length;


            vec3 L = vec3(-5, -5, -5); // tu trzeba jakis wektor swiatla globalnego

            vec3 reflection_vector = 2 * (normal * L) * normal - L;

            color diffuse = vec3(Mats[1].Diffuse[0], Mats[1].Diffuse[1], Mats[1].Diffuse[2]);
            color specular = vec3(Mats[1].Specular[0], Mats[1].Specular[1], Mats[1].Specular[2]);
            color ambient = vec3(Mats[1].Ambient[0], Mats[1].Ambient[1], Mats[1].Ambient[2]);
            float shininess = Mats[1].Shininess;
            float alpha = Mats[1].Alpha;

            color light_ambient = vec3(0.1f, 0.1f, 0.1f);
            color light_diffuse = vec3(0.1f, 0.1f, 0.1f);
            color light_specular = vec3(0.2f, 0.2f, 0.2f);
            vec3 camera_vector = rays[j * WIDTH + i].dir;

            color face_color = ambient * light_ambient + diffuse * light_diffuse * max(dotProduct_(normal, L), 0.0f) + specular * light_specular * pow(max(dotProduct_(reflection_vector, camera_vector), 0.0f), shininess) * alpha;
            //face_color = diffuse * light_diffuse * max(dotProduct_(normal, L), 0.0f);
            //printf("Color: %f, %f, %f   \n", Mats[0].Diffuse[0], Mats[0].Diffuse[1], Mats[0].Diffuse[2]);

            d_closest_normals[(j * WIDTH + i) * 3 + 0] = face_color[0];
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = face_color[1];
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = face_color[2];
        }
        //printf("%f \n", d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest]);
    }
}

__device__ void matrixVectorMul(float* matrix, float* vector, float* result) {
    float x = vector[0];
    float y = vector[1];
    float z = vector[2];
    result[0] = matrix[0] * x + matrix[4] * y + matrix[8] * z + matrix[12];
    result[1] = matrix[1] * x + matrix[5] * y + matrix[9] * z + matrix[13];
    result[2] = matrix[2] * x + matrix[6] * y + matrix[10] * z + matrix[14];
}


__global__ void Transform(float* Vertexes, int numVertices, float TranslateX, float TranslateY, float TranslateZ, float rotateX, float rotateY, float rotateZ, float scaleX, float scaleY, float scaleZ)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numVertices) 
    {
        float vertex[4] = { Vertexes[i * 3], Vertexes[i * 3 + 1], Vertexes[i * 3 + 2], 1.0f };
        float result[4];
        float transformationMatrix[16];

        transformationMatrix[0] = scaleX * (cos(rotateY)* cos(rotateZ));
        transformationMatrix[1] = scaleY * (cos(rotateZ) * sin(rotateX) * sin(rotateY) - cos(rotateX) * sin(rotateZ));
        transformationMatrix[2] = scaleZ * (cos(rotateX) * cos(rotateZ) * sin(rotateY) + sin(rotateX) * sin(rotateZ));
        transformationMatrix[3] = TranslateX;
        transformationMatrix[4] = scaleX * (cos(rotateY) * sin(rotateZ));
        transformationMatrix[5] = scaleY * (cos(rotateX) * cos(rotateZ) + sin(rotateX) * sin(rotateY) * sin(rotateZ));
        transformationMatrix[6] = scaleZ * (cos(rotateX) * sin(rotateY) * sin(rotateZ) - cos(rotateZ) * sin(rotateX));
        transformationMatrix[7] = TranslateY;
        transformationMatrix[8] = scaleX * (-sin(rotateY));
        transformationMatrix[9] = scaleY * (cos(rotateY) * sin(rotateX));
        transformationMatrix[10] = scaleZ * (cos(rotateX) * cos(rotateY));
        transformationMatrix[11] = TranslateZ;
        transformationMatrix[12] = 0.0f;
        transformationMatrix[13] = 0.0f;
        transformationMatrix[14] = 0.0f;
        transformationMatrix[15] = 1.0f;

        matrixVectorMul(transformationMatrix, vertex, result);

        Vertexes[i * 3] = result[0] + TranslateX;
        Vertexes[i * 3 + 1] = result[1] + TranslateY;
        Vertexes[i * 3 + 2] = result[2] + TranslateZ;

    }
}

__global__ void Update_normals_and_Planes(float* d_Vertices, int* d_Faces, float* d_Normals, float* d_Planes, int* d_number_of_vertices_in_one_face, int* d_normal_index_to_face, int* d_start_face_at_index, int Face_NUM, int Normals_NUM)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Face_NUM) 
    {
        int modify_normal_idx = d_normal_index_to_face[i] - 1;

        int Vertex_1_idx = d_Faces[d_start_face_at_index[i]] - 1;
        int Vertex_2_idx = d_Faces[d_start_face_at_index[i]+1] - 1;
        int Vertex_3_idx = d_Faces[d_start_face_at_index[i]+2] - 1;

        point3 Vertex_1 = point3(d_Vertices[3* Vertex_1_idx], d_Vertices[3 * Vertex_1_idx + 1], d_Vertices[3 * Vertex_1_idx + 2]);
        point3 Vertex_2 = point3(d_Vertices[3 * Vertex_2_idx], d_Vertices[3 * Vertex_2_idx + 1], d_Vertices[3 * Vertex_2_idx + 2]);
        point3 Vertex_3 = point3(d_Vertices[3 * Vertex_3_idx], d_Vertices[3 * Vertex_3_idx + 1], d_Vertices[3 * Vertex_3_idx + 2]);

        vec3 Vec_12 = vec3(Vertex_2[0] - Vertex_1[0], Vertex_2[1] - Vertex_1[1], Vertex_2[2] - Vertex_1[2]);
        vec3 Vec_13 = vec3(Vertex_3[0] - Vertex_1[0], Vertex_3[1] - Vertex_1[1], Vertex_3[2] - Vertex_1[2]);

        vec3 new_Norm =  crossProduct_(Vec_12, Vec_13);
        Plane new_Plane = Plane(new_Norm,Vertex_1);

        d_Planes[i * 4] = new_Plane.A;
        d_Planes[i * 4+1] = new_Plane.B;
        d_Planes[i * 4+2] = new_Plane.C;
        d_Planes[i * 4+3] = new_Plane.D;

        d_Normals[modify_normal_idx * 3] = new_Norm[0];
        d_Normals[modify_normal_idx * 3 + 1] = new_Norm[1];
        d_Normals[modify_normal_idx * 3 + 2] = new_Norm[2];


    }
}