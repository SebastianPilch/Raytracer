#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

__global__ void Generate_rays(ray* viewport_rays, double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index,
    int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances, float* d_closest_angles) {



    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;

    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) return;


    float aspect_ratio = float(WIDTH) / float(HEIGHT);
    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_dist = focal_length;

    vec3 w = (*camera_center - *camera_focal) / ((*camera_center - *camera_focal).length());
    vec3 u = (crossProduct_(vec3(0, 1, 0), w)) / ((crossProduct_(vec3(0, 1, 0), w)).length());
    vec3 v = crossProduct_(w,u);

    vec3 horizontal = u * viewport_width;
    vec3 vertical = v * viewport_height;
    vec3 lower_left_corner = *camera_center - horizontal / 2 - vertical / 2 - w * focal_dist;

    float u_offset = float(i) / (WIDTH - 1);
    float v_offset = float(j) / (HEIGHT - 1);
    vec3 pixel_center = lower_left_corner + horizontal * u_offset + vertical * v_offset;

/*    vec3 viewport_upper_left = *camera_center - vec3(focal_length, focal_length, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V) + *camera_focal;*/


    vec3 ray_direction = pixel_center - *camera_center;
    ray UV_ray = ray(*camera_center, ray_direction);
    viewport_rays[j * HEIGHT + i] = ray(*camera_center, ray_direction);

    __syncthreads();

    Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
    point3 intersection = UV_ray.findIntersection(plane);
    bool Is_Hitten_correct = true;
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY)
    {
        Is_Hitten_correct = false;
    }

    vec3 edge;
    size_t next_index;
    for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
    {
        size_t next_index;
        if (idx + 1 >= start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = start_face_at_index[f];
        }
        else
        {
            next_index = idx + 1;
        }
        int vertex_index = d_Faces[idx] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );

        __syncthreads();
        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) 
        {
            Is_Hitten_correct = false;
        }
        if (current_vertex[0] == 0 && current_vertex[1] == 0 && current_vertex[2] == 0)
        {
            //printf("%d", vertex_index);
        }
    }
    vec3 dis = *camera_center - intersection;

    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
        //printf("%f  %f  %f \n", intersection[0], intersection[1], intersection[2]);

    }
    else
    {
        float distance = sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = distance;

    }


    __syncthreads();
}

__global__ void Choose_closest(float* d_distances,int Face_NUM, float* d_closest_normals, float* d_Planes, Material* Mats, ray* rays, int* Mats_to_face)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;


    if (i < WIDTH && j < HEIGHT) 
    {



        color light_ambient = vec3(0.2f, 0.2f, 0.2f);
        color light_diffuse = vec3(0.7f, 0.7f, 0.7f);
        color light_specular = vec3(1.0f, 1.0f, 1.0f);
        vec3 camera_vector = rays[j * WIDTH + i].dir;

        vec3 L = vec3(-15, -15, -15); // tu trzeba jakis wektor swiatla globalnego
        //L = L / L.length();


        int closest = -1;
        float lowest_distance = INFINITY;
        float far = -1.0f;

        for (int f = 0; f < Face_NUM; f++)
        {
            if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] < lowest_distance)
            {
                closest = f;
                lowest_distance = d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f];
            }
            if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] != INFINITY && d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] > far)
            {
                far = d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f];
            }

        }

        int Mat_idx = Mats_to_face[closest] ;

        color diffuse = vec3(Mats[Mat_idx].Diffuse[0], Mats[Mat_idx].Diffuse[1], Mats[Mat_idx].Diffuse[2]);
        color specular = vec3(Mats[Mat_idx].Specular[0], Mats[Mat_idx].Specular[1], Mats[Mat_idx].Specular[2]);
        color ambient = vec3(Mats[Mat_idx].Ambient[0], Mats[Mat_idx].Ambient[1], Mats[Mat_idx].Ambient[2]);
        float shininess = Mats[Mat_idx].Shininess;
        float alpha = Mats[Mat_idx].Alpha;

        Plane plane = Plane((double)d_Planes[closest * 4], (double)d_Planes[closest * 4 + 1], (double)d_Planes[closest * 4 + 2], (double)d_Planes[closest * 4 + 3]);

        if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] == INFINITY || d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] < 0.0f)
        {
            d_closest_normals[(j * WIDTH + i) * 3 + 0] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = 0.0f;
        }
        else
        {



            vec3 normal = vec3((float)plane.A, (float)plane.B, (float)plane.C);
            //color face_color;
            float norm_length = sqrt(normal.x() * normal.x() + normal.y() * normal.y() + normal.z() * normal.z());
            normal = normal / norm_length;
            vec3 reflection_vector = 2  * dotProduct_(normal, L) * normal - L;
            color face_color = {
                ambient[0] * light_ambient[0] +
                alpha * (
                    diffuse[0] * light_diffuse[0] * max(0.0f, dotProduct_(normal, L) / L.length()) +
                    specular[0] * light_specular[0] * pow(max(0.0f, -dotProduct_(reflection_vector, camera_vector) / (reflection_vector.length() * camera_vector.length())), shininess)
                ),

                ambient[1] * light_ambient[1] +
                alpha * (
                    diffuse[1] * light_diffuse[1] * max(0.0f, dotProduct_(normal, L) / L.length()) +
                    specular[1] * light_specular[1] * pow(max(0.0f, -dotProduct_(reflection_vector, camera_vector) / (reflection_vector.length() * camera_vector.length())), shininess)
                ),

                ambient[2] * light_ambient[2] +
                alpha * (
                    diffuse[2] * light_diffuse[2] * max(0.0f, dotProduct_(normal, L) / L.length()) +
                    specular[2] * light_specular[2] * pow(max(0.0f, -dotProduct_(reflection_vector, camera_vector) / (reflection_vector.length() * camera_vector.length())), shininess)
                )
            };


            d_closest_normals[(j * WIDTH + i) * 3 + 0] = face_color[0];
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = face_color[1];
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = face_color[2];

        }
    }
}

__device__ void matrixVectorMul(float* matrix, float* vector, float* result) 
{
    float x = vector[0];
    float y = vector[1];
    float z = vector[2];
    result[0] = matrix[0] * x + matrix[4] * y + matrix[8] * z + matrix[12];
    result[1] = matrix[1] * x + matrix[5] * y + matrix[9] * z + matrix[13];
    result[2] = matrix[2] * x + matrix[6] * y + matrix[10] * z + matrix[14];
}


__global__ void Transform(float* Vertexes, int numVertices, int* Object_to_Vertex, int index, float TranslateX, float TranslateY, float TranslateZ, float rotateX, float rotateY, float rotateZ, float scaleX, float scaleY, float scaleZ)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numVertices) 
    {
        if (Object_to_Vertex[i] == index) 
        {
            float vertex[4] = { Vertexes[i * 3], Vertexes[i * 3 + 1], Vertexes[i * 3 + 2], 1.0f };
            float result[4];
            float transformationMatrix[16];

            transformationMatrix[0] = scaleX * (cos(rotateY) * cos(rotateZ));
            transformationMatrix[1] = scaleY * (cos(rotateZ) * sin(rotateX) * sin(rotateY) - cos(rotateX) * sin(rotateZ));
            transformationMatrix[2] = scaleZ * (cos(rotateX) * cos(rotateZ) * sin(rotateY) + sin(rotateX) * sin(rotateZ));
            transformationMatrix[3] = TranslateX;
            transformationMatrix[4] = scaleX * (cos(rotateY) * sin(rotateZ));
            transformationMatrix[5] = scaleY * (cos(rotateX) * cos(rotateZ) + sin(rotateX) * sin(rotateY) * sin(rotateZ));
            transformationMatrix[6] = scaleZ * (cos(rotateX) * sin(rotateY) * sin(rotateZ) - cos(rotateZ) * sin(rotateX));
            transformationMatrix[7] = TranslateY;
            transformationMatrix[8] = scaleX * (-sin(rotateY));
            transformationMatrix[9] = scaleY * (cos(rotateY) * sin(rotateX));
            transformationMatrix[10] = scaleZ * (cos(rotateX) * cos(rotateY));
            transformationMatrix[11] = TranslateZ;
            transformationMatrix[12] = 0.0f;
            transformationMatrix[13] = 0.0f;
            transformationMatrix[14] = 0.0f;
            transformationMatrix[15] = 1.0f;

            matrixVectorMul(transformationMatrix, vertex, result);

            Vertexes[i * 3] = result[0] + TranslateX;
            Vertexes[i * 3 + 1] = result[1] + TranslateY;
            Vertexes[i * 3 + 2] = result[2] + TranslateZ;
        }
    }
}

__global__ void Update_normals_and_Planes(float* d_Vertices, int* d_Faces, int* Object_to_Face, int index, float* d_Normals, float* d_Planes, int* d_number_of_vertices_in_one_face, int* d_normal_index_to_face, int* d_start_face_at_index, int Face_NUM, int Normals_NUM)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Face_NUM) 
    {
            int modify_normal_idx = d_normal_index_to_face[i] - 1;

            int Vertex_1_idx = d_Faces[d_start_face_at_index[i]] - 1;
            int Vertex_2_idx = d_Faces[d_start_face_at_index[i] + 1] - 1;
            int Vertex_3_idx = d_Faces[d_start_face_at_index[i] + 2] - 1;

            point3 Vertex_1 = point3(d_Vertices[3 * Vertex_1_idx], d_Vertices[3 * Vertex_1_idx + 1], d_Vertices[3 * Vertex_1_idx + 2]);
            point3 Vertex_2 = point3(d_Vertices[3 * Vertex_2_idx], d_Vertices[3 * Vertex_2_idx + 1], d_Vertices[3 * Vertex_2_idx + 2]);
            point3 Vertex_3 = point3(d_Vertices[3 * Vertex_3_idx], d_Vertices[3 * Vertex_3_idx + 1], d_Vertices[3 * Vertex_3_idx + 2]);

            vec3 Vec_12 = vec3(Vertex_2[0] - Vertex_1[0], Vertex_2[1] - Vertex_1[1], Vertex_2[2] - Vertex_1[2]);
            vec3 Vec_13 = vec3(Vertex_3[0] - Vertex_1[0], Vertex_3[1] - Vertex_1[1], Vertex_3[2] - Vertex_1[2]);

            vec3 new_Norm = crossProduct_(Vec_12, Vec_13);
            Plane new_Plane = Plane(new_Norm, Vertex_1);

            d_Planes[i * 4] = new_Plane.A;
            d_Planes[i * 4 + 1] = new_Plane.B;
            d_Planes[i * 4 + 2] = new_Plane.C;
            d_Planes[i * 4 + 3] = new_Plane.D;

            d_Normals[modify_normal_idx * 3] = new_Norm[0];
            d_Normals[modify_normal_idx * 3 + 1] = new_Norm[1];
            d_Normals[modify_normal_idx * 3 + 2] = new_Norm[2];
    }
}



__device__ int partition(float* data, int left, int right) {
    float pivot = data[right];
    int i = left - 1;
    for (int j = left; j < right; ++j) {
        if (data[j] < pivot) {
            ++i;
            float temp = data[i];
            data[i] = data[j];
            data[j] = temp;
        }
    }
    float temp = data[i + 1];
    data[i + 1] = data[right];
    data[right] = temp;
    return i + 1;
}

__device__ void quickSort(float* data, int left, int right) {
    if (left < right) {
        int pi = partition(data, left, right);

        quickSort(data, left, pi - 1);
        quickSort(data, pi + 1, right);
    }
}

__global__ void quickSortKernel(float* d_distances, int Face_NUM) {
    int i_ind = blockIdx.x * blockDim.x + threadIdx.x;
    int j_ind = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i_ind < WIDTH && j_ind < HEIGHT) 
    {
        int start = j_ind * Face_NUM * WIDTH + i_ind * Face_NUM;
        int end = start + Face_NUM - 1;
        quickSort(d_distances, start, end);
    }
}

