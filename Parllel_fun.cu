#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif




//std::vector<std::vector<float>> img(img_height, std::vector<float>(img_width));
//
//for (int j = 0; j < img_height; j++) {
//    for (int i = 0; i < img_width; i++) {
//        auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
//        auto ray_direction = pixel_center - camera_center;
//        ray newRay = ray(camera_center, ray_direction);
//
//        float hit_anything = 0.0;
//        for (const auto& pair : Planes_to_faces) {
//            bool hit = Face_hit(pair.second, newRay, vertices_to_faces[pair.first], vertices_coors);
//            if (hit) {
//                hit_anything += 0.2;
//            }
//
//        }
//
//        img[j][i] = hit_anything;
//    }
//}

//void saveAsBMP(const std::vector<std::vector<float>>& img, int width, int height, const std::string& filename) {
//    std::ofstream file(filename, std::ios::out | std::ios::binary);
//
//    if (!file) {
//        std::cerr << "Cannot open file: " << filename << std::endl;
//        return;
//    }
//
//    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding wymagany przez format BMP
//
//    // Nag��wek BMP
//    int filesize = 54 + (3 * width + paddingSize) * height;
//    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };
//
//    // Zapisanie nag��wka
//    file.write(fileHeader, 54);
//
//    // Zapisanie danych pikseli
//    for (int i = height - 1; i >= 0; i--) {
//        for (int j = 0; j < width; j++) {
//            unsigned char color = static_cast<unsigned char>(img[i][j] * 255); // Skalowanie warto�ci z [0, 1] do [0, 255]
//            file.put(color);
//            file.put(color);
//            file.put(color);
//        }
//        // Dodanie paddingu
//        for (int k = 0; k < paddingSize; k++) {
//            file.put(0);
//        }
//    }
//
//    file.close();
//}


__global__ void Generate_rays(ray* viewport_rays,double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes,int* start_face_at_index 
    ,int Face_NUM, int Vertex_NUM, int Normal_NUM)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    vec3 viewport_upper_left = *camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V);
    vec3 ray_direction = pixel_center - *camera_center;
    viewport_rays[j*HEIGHT + i] = ray(*camera_center , ray_direction);



}




__global__ void Face_hit()
{






};