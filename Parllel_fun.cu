#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif


//__global__ void Matrixadd(int m, int n, float* d_x, float* d_y, float* d_z)
//{
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i < m * n)
//    {
//        d_z[i] = d_x[i] + d_y[i];
//    }
//
//}


//void MatrixAddition(int N, int M, float** x, float** y, float** z)
//{
//    x[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) x[i] = x[0] + i * N;
//    y[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) y[i] = y[0] + i * N;
//    z[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) z[i] = z[0] + i * N;
//    float* d_x, *d_y, *d_z;
//    hipMalloc(&d_x, N * M * sizeof(float));
//    hipMalloc(&d_y, N * M * sizeof(float));
//    hipMalloc(&d_z, N * M * sizeof(float));
//    hipMemcpy(d_x, x[0], N * M * sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, y[0], N * M * sizeof(float), hipMemcpyHostToDevice);
//    Matrixadd << <M, N >> > (M, N, d_x, d_y, d_z);
//    hipMemcpy(z[0], d_z, N * M * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree(d_x);
//    hipFree(d_y);
//    hipFree(d_z);
//    delete[] x[0];
//    delete[] x;
//    delete[] y[0];
//    delete[] y;
//    delete[] z[0];
//    delete[] z;
//}

//int img_height = 400;
//int img_width = 400;
//
//auto focal_length = 1.0;
//auto viewport_height = 2.0;
//auto viewport_width = viewport_height * (double(img_width) / img_height);
//auto camera_center = point3(0, 0, 5);
//auto viewport_u = vec3(viewport_width, 0, 0);
//auto viewport_v = vec3(0, -viewport_height, 0);
//
//auto pixel_delta_u = viewport_u / img_width;
//auto pixel_delta_v = viewport_v / img_height;
//auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
//auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);
//
//
//std::vector<std::vector<float>> img(img_height, std::vector<float>(img_width));
//
//for (int j = 0; j < img_height; j++) {
//    for (int i = 0; i < img_width; i++) {
//        auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
//        auto ray_direction = pixel_center - camera_center;
//        ray newRay = ray(camera_center, ray_direction);
//
//        float hit_anything = 0.0;
//        for (const auto& pair : Planes_to_faces) {
//            bool hit = Face_hit(pair.second, newRay, vertices_to_faces[pair.first], vertices_coors);
//            if (hit) {
//                hit_anything += 0.2;
//            }
//
//        }
//
//        img[j][i] = hit_anything;
//    }
//}

//void saveAsBMP(const std::vector<std::vector<float>>& img, int width, int height, const std::string& filename) {
//    std::ofstream file(filename, std::ios::out | std::ios::binary);
//
//    if (!file) {
//        std::cerr << "Cannot open file: " << filename << std::endl;
//        return;
//    }
//
//    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding wymagany przez format BMP
//
//    // Nag��wek BMP
//    int filesize = 54 + (3 * width + paddingSize) * height;
//    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };
//
//    // Zapisanie nag��wka
//    file.write(fileHeader, 54);
//
//    // Zapisanie danych pikseli
//    for (int i = height - 1; i >= 0; i--) {
//        for (int j = 0; j < width; j++) {
//            unsigned char color = static_cast<unsigned char>(img[i][j] * 255); // Skalowanie warto�ci z [0, 1] do [0, 255]
//            file.put(color);
//            file.put(color);
//            file.put(color);
//        }
//        // Dodanie paddingu
//        for (int k = 0; k < paddingSize; k++) {
//            file.put(0);
//        }
//    }
//
//    file.close();
//}


__global__ void Generate_rays(ray* viewport_rays,double focal_length, point3 camera_center, point3 camera_focal)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    auto pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V);
    auto ray_direction = pixel_center - camera_center;
    viewport_rays[j*HEIGHT + i] = ray(camera_center, ray_direction);
}
