#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif


__global__ void Generate_rays(ray* viewport_rays, double focal_length, point3* camera_center, point3* camera_focal)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= WIDTH || j >= HEIGHT || z != 0) {
        return;
    }

    float aspect_ratio = float(WIDTH) / float(HEIGHT);
    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_dist = focal_length;

    vec3 w = (*camera_center - *camera_focal) / ((*camera_center - *camera_focal).length());
    vec3 u = (crossProduct_(vec3(0, 1, 0), w)) / ((crossProduct_(vec3(0, 1, 0), w)).length());
    vec3 v = crossProduct_(w, u);

    vec3 horizontal = u * viewport_width;
    vec3 vertical = v * viewport_height;
    vec3 lower_left_corner = *camera_center - horizontal / 2 - vertical / 2 - w * focal_dist;

    float u_offset = float(i) / (WIDTH - 1);
    float v_offset = float(j) / (HEIGHT - 1);
    vec3 pixel_center = lower_left_corner + horizontal * u_offset + vertical * v_offset;

    vec3 ray_direction = pixel_center - *camera_center;

    viewport_rays[j * WIDTH + i] = ray(*camera_center, ray_direction);

    __syncthreads();

}

__global__ void Update_rays(ray* viewport_rays, float* First_Intersections, int* Intersected_face_idx, int* d_normal_index_to_face, float* d_Normals)
{    
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int z = blockIdx.z * blockDim.z + threadIdx.z;

        if (i >= WIDTH || j >= HEIGHT || z != 0) 
        {
            return;
        } 


        vec3 UV_ray_dir = viewport_rays[j * WIDTH + i].dir;
        int f = Intersected_face_idx[j * WIDTH + i];
        if (f == -1) 
        {
            return;
        }
        int normal_idx = d_normal_index_to_face[f];
        vec3 normal = vec3(d_Normals[3 * normal_idx], d_Normals[3 * normal_idx + 1], d_Normals[3 * normal_idx + 2]);
        normal = normal / normal.length();
        UV_ray_dir = UV_ray_dir / UV_ray_dir.length();
        vec3 ray_direction = UV_ray_dir - 2 * dotProduct_(normal, UV_ray_dir) * normal ;

        point3 pixel_center = vec3(First_Intersections[(j * WIDTH + i) * 3], First_Intersections[(j * WIDTH + i) * 3 + 1], First_Intersections[(j * WIDTH + i) * 3 + 2]);

        viewport_rays[j * WIDTH + i] = ray(pixel_center, ray_direction);

       //printf("%f , %f , %f \n", First_Intersections[j * WIDTH + i], First_Intersections[j * WIDTH + i+1], First_Intersections[j * WIDTH + i+2]);


        __syncthreads();

}



__global__ void Generate_distances(ray* viewport_rays,point3* camera_center, float* d_intersections, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index,
    int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances, int reflections)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;

    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) 
    {
        return;
    }
    bool Is_Hitten_correct = true;

    ray UV_ray = viewport_rays[j * HEIGHT + i];

    Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
    inter_ inter_data = UV_ray.findIntersection(plane);
    point3 intersection = inter_data.intersection;
    double t = inter_data.t;


    vec3 edge;
    size_t next_index;
    for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
    {
        size_t next_index;
        if (idx + 1 >= start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = start_face_at_index[f];
        }
        else
        {
            next_index = idx + 1;
        }
        int vertex_index = d_Faces[idx] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );

        __syncthreads();
        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) 
        {
            Is_Hitten_correct = false;
        }
        if (current_vertex[0] == 0 && current_vertex[1] == 0 && current_vertex[2] == 0)
        {
            //printf("%d", vertex_index);
        }
    }
    vec3 dis;
    if (reflections == 0) 
    {
        dis = *camera_center - intersection;
    }
    else 
    {
        point3 last_inter = vec3(d_intersections[(j * WIDTH + i) * 3], d_intersections[(j * WIDTH + i) * 3 + 1], d_intersections[(j * WIDTH + i)*3 + 2]);
        dis = last_inter - intersection;

    }

    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY || t < 0)
    {
        Is_Hitten_correct = false;
    }



    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
        //printf("%f  %f  %f \n", intersection[0], intersection[1], intersection[2]);

    }
    else
    {
        float distance = sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = distance;

    }


    __syncthreads();
}

__global__ void Choose_closest(float* d_distances,int Face_NUM, float* d_colors, float* d_Planes, Material* Mats, ray* rays, int* Mats_to_face, int* Intersected_face_idx, float* d_intersections)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    

    if (i < WIDTH && j < HEIGHT && z == 1)
    {
        ray UV_ray = rays[j * WIDTH + i];


        color light_ambient = vec3(0.2f, 0.2f, 0.2f);
        color light_diffuse = vec3(0.7f, 0.7f, 0.7f);
        color light_specular = vec3(1.0f, 1.0f, 1.0f);
        vec3 camera_vector = rays[j * WIDTH + i].dir;

        vec3 L = vec3(-10, 12, 15); // tu trzeba jakis wektor swiatla globalnego
        //L = L / L.length();


        int closest = -1;
        float lowest_distance = INFINITY;
        float far = -1.0f;

        for (int f = 0; f < Face_NUM; f++)
        {
            if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] < lowest_distance && f != Intersected_face_idx[(j * WIDTH + i)])
            {
                closest = f;
                lowest_distance = d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f];
            }


        }
        if (lowest_distance == INFINITY || closest == -1)
        {
            d_colors[(j * WIDTH + i) * 3 + 0] = 0.5f;
            d_colors[(j * WIDTH + i) * 3 + 1] = 0.5f;
            d_colors[(j * WIDTH + i) * 3 + 2] = 0.5f;

            //printf("Indeks: %d, a f: %d\n", j * WIDTH + i, z);

            d_intersections[(j * WIDTH + i)*3] = INFINITY;
            d_intersections[(j * WIDTH + i) * 3 + 1] = INFINITY;
            d_intersections[(j * WIDTH + i) * 3 + 2] = INFINITY;

            return;
        }
        Plane plane = Plane((double)d_Planes[closest * 4], (double)d_Planes[closest * 4 + 1], (double)d_Planes[closest * 4 + 2], (double)d_Planes[closest * 4 + 3]);
        inter_ inter_data  = UV_ray.findIntersection(plane);
        point3 intersection = inter_data.intersection;
        double t = inter_data.t;



        d_intersections[(j * WIDTH + i) * 3] = intersection[0];
        d_intersections[(j * WIDTH + i) * 3 + 1] = intersection[1];
        d_intersections[(j * WIDTH + i) * 3 + 2] = intersection[2];
        //printf("%f , %f , %f \n", d_intersections[j * WIDTH + i], d_intersections[j * WIDTH + i+1], d_intersections[j * WIDTH + i+2]);


        Intersected_face_idx[j * WIDTH + i] = closest;

        int Mat_idx = Mats_to_face[closest];

        color diffuse = vec3(Mats[Mat_idx].Diffuse[0], Mats[Mat_idx].Diffuse[1], Mats[Mat_idx].Diffuse[2]);
        color specular = vec3(Mats[Mat_idx].Specular[0], Mats[Mat_idx].Specular[1], Mats[Mat_idx].Specular[2]);
        color ambient = vec3(Mats[Mat_idx].Ambient[0], Mats[Mat_idx].Ambient[1], Mats[Mat_idx].Ambient[2]);
        float shininess = Mats[Mat_idx].Shininess;
        float alpha = Mats[Mat_idx].Alpha;



            vec3 normal = vec3((float)plane.A, (float)plane.B, (float)plane.C);
            normal = - normal / normal.length();

            L = L / L.length();
            vec3 reflection_vector = 2 * dotProduct_(normal, L) * normal - L;
            reflection_vector = reflection_vector / reflection_vector.length();
            camera_vector = camera_vector / camera_vector.length();

            color face_color = {
                alpha * (
                    ambient[0] * light_ambient[0] +
                    diffuse[0] * light_diffuse[0] * max(0.0f, dotProduct_(normal, L)) +
                    specular[0] * light_specular[0] * pow(max(0.0f, dotProduct_(reflection_vector, camera_vector)), shininess)
                ),

                alpha * (
                    ambient[1] * light_ambient[1] +
                    diffuse[1] * light_diffuse[1] * max(0.0f, dotProduct_(normal, L)) +
                    specular[1] * light_specular[1] * pow(max(0.0f, dotProduct_(reflection_vector, camera_vector)), shininess)
                ),

                alpha * (
                    ambient[2] * light_ambient[2] +
                    diffuse[2] * light_diffuse[2] * max(0.0f, dotProduct_(normal, L)) +
                    specular[2] * light_specular[2] * pow(max(0.0f, dotProduct_(reflection_vector, camera_vector)), shininess)
                )
            };

            face_color[0] = min(1.0f, max(0.0f, face_color[0]));
            face_color[1] = min(1.0f, max(0.0f, face_color[1]));
            face_color[2] = min(1.0f, max(0.0f, face_color[2]));

                //printf("%f\n", face_color[0]);
            d_colors[(j * WIDTH + i) * 3 + 0] = face_color[0];
            d_colors[(j * WIDTH + i) * 3 + 1] = face_color[1];
            d_colors[(j * WIDTH + i) * 3 + 2] = face_color[2];



    }
}




__global__ void Add_shadows(float* d_intersections, float* d_shadows, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face, int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index, int Face_NUM, int Vertex_NUM, int Normal_NUM)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;



    if (i < WIDTH && j < HEIGHT && f < Face_NUM)
    {


        //if (d_intersections[(j * WIDTH + i) * 3] == INFINITY || d_intersections[(j * WIDTH + i) * 3 + 1] == INFINITY || d_intersections[(j * WIDTH + i) * 3 + 2] == INFINITY)
        //{
        //    return;
        //}
        ray Reverse_Light = ray(point3(d_intersections[(j * WIDTH + i) * 3], d_intersections[(j * WIDTH + i) * 3 + 1], d_intersections[(j * WIDTH + i) * 3 + 2]), vec3(-1,-1, -1));

        bool Is_Hitten_correct = true;
        Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
        inter_ inter_data = Reverse_Light.findIntersection(plane);
        point3 intersection = inter_data.intersection;
        double t = inter_data.t;

        if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
            intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY)
        {
            Is_Hitten_correct = false;
        }
        //printf("%f , %f , %f \n", d_intersections[(j * WIDTH + i) * 3], d_intersections[(j * WIDTH + i) * 3 + 1], d_intersections[(j * WIDTH + i) * 3 + 2]);

        vec3 edge;
        size_t next_index;
        for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
        {
            size_t next_index;
            if (idx + 1 >= start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
            {
                next_index = start_face_at_index[f];
            }
            else
            {
                next_index = idx + 1;
            }
            int vertex_index = d_Faces[idx] - 1;
            int vertex_next = d_Faces[next_index] - 1;

            point3 current_vertex = point3(
                (double)d_Vertices[3 * vertex_index],
                (double)d_Vertices[3 * vertex_index + 1],
                (double)d_Vertices[3 * vertex_index + 2]
            );

            point3 next_vertex = point3(
                (double)d_Vertices[3 * vertex_next],
                (double)d_Vertices[3 * vertex_next + 1],
                (double)d_Vertices[3 * vertex_next + 2]
            );

            __syncthreads();
            edge = next_vertex - current_vertex;
            vec3 vp = intersection - current_vertex;
            vec3 n = crossProduct_(edge, vp);
            vec3 normal = vec3(plane.A, plane.B, plane.C);
            if (dotProduct_(n, normal) < 0)
            {
                Is_Hitten_correct = false;
            }
        }


        if(Is_Hitten_correct && t > 0 )
        {
            d_shadows[(j * WIDTH + i)*3] = 1;
            d_shadows[(j * WIDTH + i)*3 + 1] = 1;
            d_shadows[(j * WIDTH + i)*3 + 2] = 1;

        }


        __syncthreads();
    }
}















__device__ void matrixVectorMul(float* matrix, float* vector, float* result) 
{
    float x = vector[0];
    float y = vector[1];
    float z = vector[2];
    result[0] = matrix[0] * x + matrix[4] * y + matrix[8] * z + matrix[12];
    result[1] = matrix[1] * x + matrix[5] * y + matrix[9] * z + matrix[13];
    result[2] = matrix[2] * x + matrix[6] * y + matrix[10] * z + matrix[14];
}


__global__ void Transform(float* Vertexes, int numVertices, int* Object_to_Vertex, int index, float TranslateX, float TranslateY, float TranslateZ, float rotateX, float rotateY, float rotateZ, float scaleX, float scaleY, float scaleZ)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numVertices) 
    {
        if (Object_to_Vertex[i] == index) 
        {
            float vertex[4] = { Vertexes[i * 3], Vertexes[i * 3 + 1], Vertexes[i * 3 + 2], 1.0f };
            float result[4];
            float transformationMatrix[16];

            transformationMatrix[0] = scaleX * (cos(rotateY) * cos(rotateZ));
            transformationMatrix[1] = scaleY * (cos(rotateZ) * sin(rotateX) * sin(rotateY) - cos(rotateX) * sin(rotateZ));
            transformationMatrix[2] = scaleZ * (cos(rotateX) * cos(rotateZ) * sin(rotateY) + sin(rotateX) * sin(rotateZ));
            transformationMatrix[3] = TranslateX;
            transformationMatrix[4] = scaleX * (cos(rotateY) * sin(rotateZ));
            transformationMatrix[5] = scaleY * (cos(rotateX) * cos(rotateZ) + sin(rotateX) * sin(rotateY) * sin(rotateZ));
            transformationMatrix[6] = scaleZ * (cos(rotateX) * sin(rotateY) * sin(rotateZ) - cos(rotateZ) * sin(rotateX));
            transformationMatrix[7] = TranslateY;
            transformationMatrix[8] = scaleX * (-sin(rotateY));
            transformationMatrix[9] = scaleY * (cos(rotateY) * sin(rotateX));
            transformationMatrix[10] = scaleZ * (cos(rotateX) * cos(rotateY));
            transformationMatrix[11] = TranslateZ;
            transformationMatrix[12] = 0.0f;
            transformationMatrix[13] = 0.0f;
            transformationMatrix[14] = 0.0f;
            transformationMatrix[15] = 1.0f;

            matrixVectorMul(transformationMatrix, vertex, result);

            Vertexes[i * 3] = result[0] + TranslateX;
            Vertexes[i * 3 + 1] = result[1] + TranslateY;
            Vertexes[i * 3 + 2] = result[2] + TranslateZ;
        }
    }
}

__global__ void Update_normals_and_Planes(float* d_Vertices, int* d_Faces, int* Object_to_Face, int index, float* d_Normals, float* d_Planes, int* d_number_of_vertices_in_one_face, int* d_normal_index_to_face, int* d_start_face_at_index, int Face_NUM, int Normals_NUM)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Face_NUM) 
    {
            int modify_normal_idx = d_normal_index_to_face[i] - 1;

            int Vertex_1_idx = d_Faces[d_start_face_at_index[i]] - 1;
            int Vertex_2_idx = d_Faces[d_start_face_at_index[i] + 1] - 1;
            int Vertex_3_idx = d_Faces[d_start_face_at_index[i] + 2] - 1;

            point3 Vertex_1 = point3(d_Vertices[3 * Vertex_1_idx], d_Vertices[3 * Vertex_1_idx + 1], d_Vertices[3 * Vertex_1_idx + 2]);
            point3 Vertex_2 = point3(d_Vertices[3 * Vertex_2_idx], d_Vertices[3 * Vertex_2_idx + 1], d_Vertices[3 * Vertex_2_idx + 2]);
            point3 Vertex_3 = point3(d_Vertices[3 * Vertex_3_idx], d_Vertices[3 * Vertex_3_idx + 1], d_Vertices[3 * Vertex_3_idx + 2]);

            vec3 Vec_12 = vec3(Vertex_2[0] - Vertex_1[0], Vertex_2[1] - Vertex_1[1], Vertex_2[2] - Vertex_1[2]);
            vec3 Vec_13 = vec3(Vertex_3[0] - Vertex_1[0], Vertex_3[1] - Vertex_1[1], Vertex_3[2] - Vertex_1[2]);

            vec3 new_Norm = crossProduct_(Vec_12, Vec_13);
            Plane new_Plane = Plane(new_Norm, Vertex_1);

            d_Planes[i * 4] = new_Plane.A;
            d_Planes[i * 4 + 1] = new_Plane.B;
            d_Planes[i * 4 + 2] = new_Plane.C;
            d_Planes[i * 4 + 3] = new_Plane.D;

            d_Normals[modify_normal_idx * 3] = new_Norm[0];
            d_Normals[modify_normal_idx * 3 + 1] = new_Norm[1];
            d_Normals[modify_normal_idx * 3 + 2] = new_Norm[2];
    }
}

























__device__ int partition(float* data, int left, int right) {
    float pivot = data[right];
    int i = left - 1;
    for (int j = left; j < right; ++j) {
        if (data[j] < pivot) {
            ++i;
            float temp = data[i];
            data[i] = data[j];
            data[j] = temp;
        }
    }
    float temp = data[i + 1];
    data[i + 1] = data[right];
    data[right] = temp;
    return i + 1;
}

__device__ void quickSort(float* data, int left, int right) {
    if (left < right) {
        int pi = partition(data, left, right);

        quickSort(data, left, pi - 1);
        quickSort(data, pi + 1, right);
    }
}

__global__ void quickSortKernel(float* d_distances, int Face_NUM) {
    int i_ind = blockIdx.x * blockDim.x + threadIdx.x;
    int j_ind = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i_ind < WIDTH && j_ind < HEIGHT) 
    {
        int start = j_ind * Face_NUM * WIDTH + i_ind * Face_NUM;
        int end = start + Face_NUM - 1;
        quickSort(d_distances, start, end);
    }
}

