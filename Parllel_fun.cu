#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

__global__ void Generate_rays(ray* viewport_rays, double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index,
    int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances, float* d_closest_angles) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;
    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) return;

    vec3 viewport_upper_left = *camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V);
    vec3 ray_direction = pixel_center - *camera_center;
    ray UV_ray = ray(*camera_center, ray_direction);
    viewport_rays[j * HEIGHT + i] = ray(*camera_center, ray_direction);

    __syncthreads();

    Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
    point3 intersection = UV_ray.findIntersection(plane);
    bool Is_Hitten_correct = true;
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY)
    {
        Is_Hitten_correct = false;
    }

    vec3 edge;
    size_t next_index;

    for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
    {
        size_t next_index;
        if (idx + 1 > start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = idx + 1;
        }
        else
        {
            next_index = start_face_at_index[f];
        }
        int vertex_index = d_Faces[idx] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );

        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) {
            Is_Hitten_correct = false;
        }
    }
    vec3 dis = *camera_center - intersection;
    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
        d_closest_angles[(j * WIDTH + i) * 3 + 0] = 0.0f;
        d_closest_angles[(j * WIDTH + i) * 3 + 1] = 0.0f;
        d_closest_angles[(j * WIDTH + i) * 3 + 2] = 0.0f;
    }
    else
    {
        float distance = sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = distance;

        vec3 normal = vec3((float)plane.A, (float)plane.B, (float)plane.C);
        float norm_length = sqrt(normal.x() * normal.x() + normal.y() * normal.y() + normal.z() * normal.z());
        normal = normal / norm_length;

        d_closest_angles[(j * WIDTH + i) * 3 + 0] = (normal.x() + 1.0f) * 0.5f;
        d_closest_angles[(j * WIDTH + i) * 3 + 1] = (normal.y() + 1.0f) * 0.5f;
        d_closest_angles[(j * WIDTH + i) * 3 + 2] = (normal.z() + 1.0f) * 0.5f;
    }
    __syncthreads();
}
