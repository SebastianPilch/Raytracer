#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif




//std::vector<std::vector<float>> img(img_height, std::vector<float>(img_width));
//
//for (int j = 0; j < img_height; j++) {
//    for (int i = 0; i < img_width; i++) {
//        auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
//        auto ray_direction = pixel_center - camera_center;
//        ray newRay = ray(camera_center, ray_direction);
//
//        float hit_anything = 0.0;
//        for (const auto& pair : Planes_to_faces) {
//            bool hit = Face_hit(pair.second, newRay, vertices_to_faces[pair.first], vertices_coors);
//            if (hit) {
//                hit_anything += 0.2;
//            }
//
//        }
//
//        img[j][i] = hit_anything;
//    }
//}

//void saveAsBMP(const std::vector<std::vector<float>>& img, int width, int height, const std::string& filename) {
//    std::ofstream file(filename, std::ios::out | std::ios::binary);
//
//    if (!file) {
//        std::cerr << "Cannot open file: " << filename << std::endl;
//        return;
//    }
//
//    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding wymagany przez format BMP
//
//    // Nag��wek BMP
//    int filesize = 54 + (3 * width + paddingSize) * height;
//    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };
//
//    // Zapisanie nag��wka
//    file.write(fileHeader, 54);
//
//    // Zapisanie danych pikseli
//    for (int i = height - 1; i >= 0; i--) {
//        for (int j = 0; j < width; j++) {
//            unsigned char color = static_cast<unsigned char>(img[i][j] * 255); // Skalowanie warto�ci z [0, 1] do [0, 255]
//            file.put(color);
//            file.put(color);
//            file.put(color);
//        }
//        // Dodanie paddingu
//        for (int k = 0; k < paddingSize; k++) {
//            file.put(0);
//        }
//    }
//
//    file.close();
//}


__global__ void Generate_rays(ray* viewport_rays,double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes,int* start_face_at_index 
    ,int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;

    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) return;

    vec3 viewport_upper_left = *camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V);
    vec3 ray_direction = pixel_center - *camera_center;
    ray UV_ray = ray(*camera_center, ray_direction);
    viewport_rays[j * HEIGHT + i] = ray(*camera_center, ray_direction);
    
    __syncthreads();

    
    Plane plane = Plane((double)d_Planes[f*4], (double)d_Planes[f * 4+1], (double)d_Planes[f * 4+2], (double)d_Planes[f * 4+3]);
    point3 intersection = UV_ray.findIntersection(plane);
    bool Is_Hitten_correct = true;
 
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY) 
    {
        Is_Hitten_correct = false;
    }
    
    vec3 edge;
    size_t next_index;
    for (size_t i = start_face_at_index[f]; i < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++i) 
    {
        size_t next_index;
        if (i + 1 > start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = i + 1;
        }
        else
        {
            next_index = start_face_at_index[f];
        }
        int vertex_index = d_Faces[i] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );


        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) {
            Is_Hitten_correct = false;
        }
    }
    vec3 dis = *camera_center - intersection;
    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
    }
    else
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] =(float)plane.A;//sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
    }
    __syncthreads();
//


        //Face_hit << <Cuda_Blocks, Threads_in_one_block >> > (d_Planes, UV_ray, d_number_of_vertices_in_one_face, d_Faces, d_Vertices, Face_NUM, start_face_at_index);




}


