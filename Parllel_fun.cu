#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"

using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

__global__ void Generate_rays(ray* viewport_rays, double focal_length, point3* camera_center,
    point3* camera_focal, int* d_normal_index_to_face, int* d_number_of_vertices_in_one_face,
    int* d_Faces, float* d_Vertices, float* d_Normals, float* d_Planes, int* start_face_at_index,
    int Face_NUM, int Vertex_NUM, int Normal_NUM, float* d_distances, float* d_closest_angles) {



    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int f = blockIdx.z * blockDim.z + threadIdx.z;

    if (f >= Face_NUM || i >= WIDTH || j >= HEIGHT) return;


    float aspect_ratio = float(WIDTH) / float(HEIGHT);
    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_dist = focal_length;

    vec3 w = (*camera_center - *camera_focal) / ((*camera_center - *camera_focal).length());
    vec3 u = (crossProduct_(vec3(0, 1, 0), w)) / ((crossProduct_(vec3(0, 1, 0), w)).length());
    vec3 v = crossProduct_(w,u);

    vec3 horizontal = u * viewport_width;
    vec3 vertical = v * viewport_height;
    vec3 lower_left_corner = *camera_center - horizontal / 2 - vertical / 2 - w * focal_dist;
    float u_offset = float(i) / (WIDTH - 1);
    float v_offset = float(j) / (HEIGHT - 1);
    vec3 pixel_center = lower_left_corner + horizontal * u_offset + vertical * v_offset;











    //vec3 viewport_upper_left = *camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    //vec3 pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    //vec3 pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V) + *camera_focal;


    vec3 ray_direction = pixel_center - *camera_center;
    ray UV_ray = ray(*camera_center, ray_direction);
    viewport_rays[j * HEIGHT + i] = ray(*camera_center, ray_direction);

    __syncthreads();

    Plane plane = Plane((double)d_Planes[f * 4], (double)d_Planes[f * 4 + 1], (double)d_Planes[f * 4 + 2], (double)d_Planes[f * 4 + 3]);
    point3 intersection = UV_ray.findIntersection(plane);
    bool Is_Hitten_correct = true;
    if (intersection[0] == INFINITY || intersection[0] == -INFINITY || intersection[1] == INFINITY ||
        intersection[1] == -INFINITY || intersection[2] == INFINITY || intersection[2] == -INFINITY)
    {
        Is_Hitten_correct = false;
    }

    vec3 edge;
    size_t next_index;
    for (size_t idx = start_face_at_index[f]; idx < start_face_at_index[f] + d_number_of_vertices_in_one_face[f]; ++idx)
    {
        size_t next_index;
        if (idx + 1 >= start_face_at_index[f] + d_number_of_vertices_in_one_face[f])
        {
            next_index = start_face_at_index[f];
        }
        else
        {
            next_index = idx + 1;
        }
        int vertex_index = d_Faces[idx] - 1;
        int vertex_next = d_Faces[next_index] - 1;

        point3 current_vertex = point3(
            (double)d_Vertices[3 * vertex_index],
            (double)d_Vertices[3 * vertex_index + 1],
            (double)d_Vertices[3 * vertex_index + 2]
        );

        point3 next_vertex = point3(
            (double)d_Vertices[3 * vertex_next],
            (double)d_Vertices[3 * vertex_next + 1],
            (double)d_Vertices[3 * vertex_next + 2]
        );

        

        edge = next_vertex - current_vertex;
        vec3 vp = intersection - current_vertex;
        vec3 n = crossProduct_(edge, vp);
        vec3 normal = vec3(plane.A, plane.B, plane.C);
        if (dotProduct_(n, normal) < 0) 
        {
            Is_Hitten_correct = false;
        }
    }
    vec3 dis = *camera_center - intersection;

    if (!Is_Hitten_correct)
    {
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = INFINITY;
        //printf("%f  %f  %f \n", intersection[0], intersection[1], intersection[2]);

    }
    else
    {
        float distance = sqrt(dis[0] * dis[0] + dis[1] * dis[1] + dis[2] * dis[2]);
        d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] = distance;

    }


    __syncthreads();
}

__global__ void Choose_closest(float* d_distances,int Face_NUM, float* d_closest_normals, float* d_Planes)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < WIDTH && j < HEIGHT) {
        int closest = -1;
        float lowest_distance = INFINITY;

        for (int f = 0; f < Face_NUM; f++)
        {
            if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f] < lowest_distance)
            {
                closest = f;
                lowest_distance = d_distances[j * Face_NUM * WIDTH + i * Face_NUM + f];
            }
        }

        Plane plane = Plane((double)d_Planes[closest * 4], (double)d_Planes[closest * 4 + 1], (double)d_Planes[closest * 4 + 2], (double)d_Planes[closest * 4 + 3]);

        if (d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] == INFINITY || d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest] < 0.0f)
        {
            d_closest_normals[(j * WIDTH + i) * 3 + 0] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = 0.0f;
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = 0.0f;
        }
        else
        {
            vec3 normal = vec3((float)plane.A, (float)plane.B, (float)plane.C);
            float norm_length = sqrt(normal.x() * normal.x() + normal.y() * normal.y() + normal.z() * normal.z());
            normal = normal / norm_length;

            d_closest_normals[(j * WIDTH + i) * 3 + 0] = (normal.x() + 1.0f) * 0.5f;
            d_closest_normals[(j * WIDTH + i) * 3 + 1] = (normal.y() + 1.0f) * 0.5f;
            d_closest_normals[(j * WIDTH + i) * 3 + 2] = (normal.z() + 1.0f) * 0.5f;
        }
        //printf("%f \n", d_distances[j * Face_NUM * WIDTH + i * Face_NUM + closest]);
    }
}
