#include "hip/hip_runtime.h"
#include "vec3.cuh"
#include <iostream>

__host__ __device__ vec3::vec3() : e{ 0, 0, 0 } {}

__host__ __device__ vec3::vec3(double e0, double e1, double e2) : e{ e0, e1, e2 } {}

__host__ __device__ double vec3::x() const { return e[0]; }
__host__ __device__ double vec3::y() const { return e[1]; }
__host__ __device__ double vec3::z() const { return e[2]; }

__host__ __device__ vec3 vec3::operator-() const { return vec3(-e[0], -e[1], -e[2]); }
__host__ __device__ double vec3::operator[](int i) const { return e[i]; }
__host__ __device__ double& vec3::operator[](int i) { return e[i]; }

__host__ __device__ vec3& vec3::operator+=(const vec3& v) {
    e[0] += v.e[0];
    e[1] += v.e[1];
    e[2] += v.e[2];
    return *this;
}

__host__ __device__ vec3& vec3::operator*=(double t) {
    e[0] *= t;
    e[1] *= t;
    e[2] *= t;
    return *this;
}

__host__ __device__ vec3& vec3::operator/=(double t) {
    return *this *= 1 / t;
}

__host__ __device__ double vec3::length() const {
    return sqrt(length_squared());
}

__host__ __device__ double vec3::length_squared() const {
    return e[0] * e[0] + e[1] * e[1] + e[2] * e[2];
}

__host__ __device__ std::ostream& operator<<(std::ostream& out, const vec3& v) {
    return out << v.e[0] << ' ' << v.e[1] << ' ' << v.e[2];
}

__host__ __device__ inline vec3 operator+(const vec3& u, const vec3& v) {
    return vec3(u.e[0] + v.e[0], u.e[1] + v.e[1], u.e[2] + v.e[2]);
}

__host__ __device__ inline vec3 operator-(const vec3& u, const vec3& v) {
    return vec3(u.e[0] - v.e[0], u.e[1] - v.e[1], u.e[2] - v.e[2]);
}

__host__ __device__ inline vec3 operator*(const vec3& u, const vec3& v) {
    return vec3(u.e[0] * v.e[0], u.e[1] * v.e[1], u.e[2] * v.e[2]);
}

__host__ __device__ inline vec3 operator*(double t, const vec3& v) {
    return vec3(t * v.e[0], t * v.e[1], t * v.e[2]);
}

__host__ __device__ inline vec3 operator*(const vec3& v, double t) {
    return t * v;
}

__host__ __device__ inline vec3 operator/(const vec3& v, double t) {
    return (1 / t) * v;
}

__host__ __device__ inline double dot(const vec3& u, const vec3& v) {
    return u.e[0] * v.e[0]
        + u.e[1] * v.e[1]
        + u.e[2] * v.e[2];
}

__host__ __device__ inline vec3 cross(const vec3& u, const vec3& v) {
    return vec3(u.e[1] * v.e[2] - u.e[2] * v.e[1],
        u.e[2] * v.e[0] - u.e[0] * v.e[2],
        u.e[0] * v.e[1] - u.e[1] * v.e[0]);
}

__host__ __device__ inline vec3 unit_vector(const vec3& v) {
    return v / v.length();
}

__host__ __device__ inline vec3 crossProduct_(const vec3& a, const vec3& b) {
    return vec3(a.y() * b.z() - a.z() * b.y(), a.z() * b.x() - a.x() * b.z(), a.x() * b.y() - a.y() * b.x());
}

__host__ __device__ inline double dotProduct_(const vec3& a, const vec3& b) {
    return a.x() * b.x() + a.y() * b.y() + a.z() * b.z();
}

__host__ __device__ Plane::Plane() : A(0), B(0), C(0), D(0) {}

__host__ __device__ Plane::Plane(double _A, double _B, double _C, double _D) : A(_A), B(_B), C(_C), D(_D) {}

__host__ __device__ Plane::Plane(const vec3& normal, const point3& point_on_face) {
    A = normal.x();
    B = normal.y();
    C = normal.z();
    D = -(normal.x() * point_on_face.x() + normal.y() * point_on_face.y() + normal.z() * point_on_face.z());
}

__host__ __device__ Plane::Plane(const Plane& other) : A(other.A), B(other.B), C(other.C), D(other.D) {}

__host__ __device__ Plane::Plane(Plane&& other) noexcept : A(std::exchange(other.A, 0)),
B(std::exchange(other.B, 0)),
C(std::exchange(other.C, 0)),
D(std::exchange(other.D, 0)) {}

__host__ __device__ Plane& Plane::operator=(const Plane& other) {
    if (this != &other) {
        A = other.A;
        B = other.B;
        C = other.C;
        D = other.D;
    }
    return *this;
}

__host__ __device__ std::ostream& operator<<(std::ostream& out, const Plane& v) {
    if (abs(v.A) != 1 && abs(v.A) != 0) { out << v.A; }
    if (abs(v.A) != 0 && v.A > 0) { out << "x"; }
    if (abs(v.A) != 0 && v.A < 0) { out << "-x"; }
    if (abs(v.A) != 0 && abs(v.B) != 0 && v.B > 0) { out << "+"; }
    if (abs(v.B) != 0 && v.B < 0) { out << ""; }
    if (abs(v.B) != 1 && abs(v.B) != 0) { out << v.B; }
    if (abs(v.B) != 0 && v.B > 0) { out << "y"; }
    if (abs(v.B) != 0 && v.B < 0) { out << "-y"; }
    if (abs(v.A) != 0 && abs(v.B) != 0 && abs(v.C) != 0 && v.C > 0) { out << "+"; }
    if (abs(v.C) != 0 && v.C < 0) { out << ""; }
    if (abs(v.C) != 1 && abs(v.C) != 0) { out << v.C; }
    if (abs(v.C) != 0 && v.C > 0) { out << "z"; }
    if (abs(v.C) != 0 && v.C < 0) { out << "-z"; }
    if (abs(v.D) != 0 && v.D > 0) { out << "+"; }
    if (abs(v.D) != 0 && v.D < 0) { out << ""; }
    if (abs(v.D) != 0) { out << v.D << " = 0"; }
    return out;
}

__host__ __device__ Vector::Vector(float _x, float _y, float _z) : x(_x), y(_y), z(_z) {}
__host__ __device__ Vector::Vector() : x(0.0f), y(0.0f), z(0.0f) {}

__global__ void MyKernel(Vector* vectors, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        vectors[idx] = Vector(1.0f, 2.0f, 3.0f); 
    }
    vec3 x = vec3();
}
__host__ void printVectors(Vector* vectors, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << "x: " << vectors[i].x << ", y: " << vectors[i].y << ", z: " << vectors[i].z << std::endl;
    }
}