#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
#include "Material.cuh"
#include<cmath>

void saveAsBMP(ray** img, int width, int height, const std::string& filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        return;
    }

    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding required by BMP format

    // BMP header
    int filesize = 54 + (3 * width + paddingSize) * height;
    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };

    // Write header
    file.write(fileHeader, 54);

    // Write pixel data
    for (int i = height - 1; i >= 0; i--) {
        for (int j = 0; j < width; j++) {
            float focal_len = 5.0f;
            unsigned char color = static_cast<unsigned char>(focal_len / (sqrt(img[i][j].dir.x() * img[i][j].dir.x() + img[i][j].dir.y() * img[i][j].dir.y())) / 4); // Scale value from [0, 1] to [0, 255]
            file.put(color);
            file.put(color);
            file.put(color);
        }

        // Add padding
        for (int k = 0; k < paddingSize; k++) {
            file.put(0);
        }
    }

    file.close();
}


void saveAsBMP2(float* angles, int width, int height, const std::string& filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        return;
    }

    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding required by BMP format

    // BMP header
    int filesize = 54 + (3 * width + paddingSize) * height;
    char fileHeader[54] = {
        'B', 'M',                         // Signature
        static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), // File size
        0,0, 0,0,                         // Reserved
        54,0,0,0,                         // File offset to pixel array
        40,0,0,0,                         // DIB header size
        static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), // Width
        static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), // Height
        1,0,                              // Planes
        24,0,                             // Bits per pixel
        0,0,0,0,                          // Compression
        0,0,0,0,                          // Image size (can be 0 for uncompressed)
        0,0,0,0,                          // X pixels per meter (unused)
        0,0,0,0,                          // Y pixels per meter (unused)
        0,0,0,0,                          // Total colors (0 means default)
        0,0,0,0                           // Important colors (0 means all are important)
    };

    // Write header
    file.write(fileHeader, 54);

    // Write pixel data
    for (int i = height - 1; i >= 0; i--) {
        for (int j = 0; j < width; j++) {
            // Retrieve RGB values from angles array
            float red = angles[(i * width + j) * 3 + 0];
            float green = angles[(i * width + j) * 3 + 1];
            float blue = angles[(i * width + j) * 3 + 2];

            unsigned char r = static_cast<unsigned char>(red * 255.99f);
            unsigned char g = static_cast<unsigned char>(green * 255.99f);
            unsigned char b = static_cast<unsigned char>(blue * 255.99f);

            file.put(b); // Blue channel
            file.put(g); // Green channel
            file.put(r); // Red channel
        }

        // Add padding
        for (int k = 0; k < paddingSize; k++) {
            file.put(0);
        }
    }

    file.close();
}


int main() {

    ///////////////////////////////////////////////
    //
    //wczytanie obiektów .obj
    //
    //
    //
    //
    //////////////////////////////////////////////////


    //int vert_num1 = 3;
    //int face_num1 = 1;
    //int normal_num1 = 1;

    //Pointer_storage Kostka = GetDataFromObj(vert_num1, face_num1, normal_num1, "../../../helpers/untitled.obj");

    //int vert_num2 = 3;
    //int face_num2 = 1;
    //int normal_num2 = 1;

    //Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2, face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");
    int object_couter = 0;
    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    //Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");
    Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, object_couter, "../../../helpers/complete_scene.obj");

    //cout << endl << endl << "Kostka" << endl << endl;

    //cout << endl << endl << "Kostka" << endl << endl;
    //cout << endl << endl << "Kostka" << endl << endl;






    //Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    //cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    //Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    //cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;
    //Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //cout << endl << endl << "Testowanie promieni";





    ///////////////////////////////////////////////
    //
    //wybór obiektu
    //
    //
    //
    //
    //////////////////////////////////////////////////
    int Vert_NUM = vert_num3;
    int Face_NUM = face_num3;
    int Normal_NUM = normal_num3;
    Pointer_storage liczony_objekt = pociety_walec;
    ///////////////////////////////////////////////
    //
    //przepisanie wskaźników
    //
    //
    //
    //
    //////////////////////////////////////////////////
    float** Planes = new float* [Face_NUM];
    Planes[0] = new float[Face_NUM * 4];

    float** Verticies = new float* [Vert_NUM];
    Verticies[0] = new float[Vert_NUM * 3 ];

    float** Normals = new float* [Normal_NUM];
    Normals[0] = new float[Normal_NUM * 3];

    float* Distances = new float[WIDTH * HEIGHT * Face_NUM];
    float* ClosestNormals = new float[WIDTH * HEIGHT * Face_NUM * 3];

    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;

    int* start_face_at_index = new int[Face_NUM];

    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) {
        Length_to_Allocate_Faces += number_of_vertices_in_one_face[i];
    }

    int** Faces = new int* [Face_NUM];
    Faces[0] = new int[Length_to_Allocate_Faces];

    int current_index = 0;
    for (int i = 1; i < Face_NUM + 1; i++) {
        Faces[i] = Faces[0] + current_index + number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i-1] = current_index;
        current_index += number_of_vertices_in_one_face[i - 1];
        Planes[i] = Planes[0] + i * 4;
        //cout << i-1 << "  :  " << start_face_at_index[i-1] << endl;
    }

    for (int i = 1; i < Vert_NUM; i++) {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++) {
        Normals[i] = Normals[0] + i * 3;
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < 4; j++)
            Planes[i][j] = liczony_objekt.Planes[i][j];
    }
    for (int i = 0; i < Vert_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Verticies[i][j] = liczony_objekt.Vertices[i][j];
    }
    for (int i = 0; i < Normal_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Normals[i][j] = liczony_objekt.Normals[i][j];
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < liczony_objekt.Face_size[i]; j++) {
            Faces[i][j] = liczony_objekt.Faces[i][j];
        }
    }

    ///////////////////////////////////////////////
    //
    //utworzenie materiałów
    //
    //
    //
    //
    //////////////////////////////////////////////////
    Material* Materials = new Material[3];
    Materials[0] = Material( 0.5f, 0.6f, 0.7f , 0.3f, 0.4f, 0.5f , 0.1f, 0.2f, 0.3f , 1.0f, 0.0f);
    Materials[1] = Material(0.7f, 0.2f, 0.2f, 0.3f, 0.4f, 0.5f, 0.1f, 0.2f, 0.3f, 1.0f, 5.0f);
    Materials[2] = Material(0.5f, 0.6f, 0.7f, 0.3f, 0.4f, 0.5f, 0.1f, 0.2f, 0.3f, 0.7f, 0.0f);
    ///////////////////////////////////////////////
    //
    //Alokacja i przekazanie danych do karty
    //
    //
    //
    //
    //////////////////////////////////////////////////

    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;
    float* d_closest_normals;
    Material* d_Materials;

    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    hipMalloc(&d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float));
    hipMalloc(&d_closest_normals, WIDTH * HEIGHT * Face_NUM * 3 * sizeof(float));

    hipMemcpy(d_Faces, Faces[0], Length_to_Allocate_Faces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Planes, Planes[0], 4 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Normals, Normals[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Vertices, Verticies[0], 3 * Vert_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_vertices_in_one_face, number_of_vertices_in_one_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_normal_index_to_face, normal_index_to_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_face_at_index, start_face_at_index, Face_NUM * sizeof(int), hipMemcpyHostToDevice);

    ///////////////////////////////////////////////
    //
    //Tranformacja obiektu rotacja/skala/przesunięcie
    //
    //
    //
    //
    //////////////////////////////////////////////////

    int threadsPerBlock = 256;
    int blocksPerGrid = (Vert_NUM + threadsPerBlock - 1) / threadsPerBlock;
    float TranslateX = 0.0f;
    float TranslateY = 0.0f;
    float TranslateZ = 0.0f;
    float rotateX = 0.0f;
    float rotateY = 23.0f;
    float rotateZ = 23.0f;
    float scaleX = 1.0f;
    float scaleY = 1.0f;
    float scaleZ = 1.0f;


    Transform<<<blocksPerGrid, threadsPerBlock >>>(d_Vertices, Vert_NUM,  TranslateX,  TranslateY,  TranslateZ,  rotateX,  rotateY,  rotateZ,  scaleX,  scaleY,  scaleZ);
    //hipMemcpy(Verticies, d_Vertices, Vert_NUM * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    threadsPerBlock = 256;
    blocksPerGrid = (Face_NUM + threadsPerBlock - 1) / threadsPerBlock;
    Update_normals_and_Planes << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, d_Faces, d_Normals, d_Planes, d_number_of_vertices_in_one_face, d_normal_index_to_face, d_start_face_at_index,Face_NUM,Normal_NUM);
    hipDeviceSynchronize();


    ///////////////////////////////////////////////
    //
    //  Wyznaczanie Promieni, uderzenia i dystanse
    //
    //
    //
    //
    //////////////////////////////////////////////////

    double focal_length = 10;
    point3 h_camera_center(45.0, 45.0, 45.0);
    point3 h_camera_focal(1.0,1.0, 1.0);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);

    dim3 blockDim(8, 8, 8);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    Generate_rays << <gridDim, blockDim >> > (d_ray, focal_length, d_camera_center, d_camera_focal, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, d_closest_normals);
    hipDeviceSynchronize();

    cout << endl << "sort start" << endl;

    //quickSortKernel << <gridDim, blockDim >> > (d_distances, Face_NUM);

    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipMemcpy(Distances, d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float), hipMemcpyDeviceToHost);


    ///////////////////////////////////////////////
    //
    // dobór najbliższej ściany z wektora dystansów
    //
    //
    //
    //
    //////////////////////////////////////////////////

    const int BLOCK_SIZE_X = 16;
    const int BLOCK_SIZE_Y = 16;
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    int gridSizeX = (WIDTH + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
    int gridSizeY = (HEIGHT + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
    dim3 dimGrid(gridSizeX, gridSizeY);

    hipMalloc(&d_Materials,3 * sizeof(Material));
    hipMemcpy(d_distances, Distances, WIDTH * HEIGHT * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Materials, Materials, 3 * sizeof(Material), hipMemcpyHostToDevice);


    Choose_closest <<< gridDim, blockDim >>> (d_distances, Face_NUM,d_closest_normals,d_Planes,d_Materials,d_ray);
    hipDeviceSynchronize();

    hipMemcpy(ClosestNormals, d_closest_normals, WIDTH * HEIGHT * Face_NUM * 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    hipFree(d_closest_normals);

    saveAsBMP(h_ray, WIDTH, HEIGHT, "result_image.bmp");
    saveAsBMP2(ClosestNormals, WIDTH, HEIGHT, "normals_image.bmp");

    free(h_ray[0]);
    free(h_ray);
    delete[] ClosestNormals;

    return 0;
}

