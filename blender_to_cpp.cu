
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");

    cout << "XDD" << endl;
    cout << endl << endl << " Linia przed cuda";


    const int size = 10;
    Vector* d_vectors;
    vec3* d_z = (vec3*)malloc(size * sizeof(vec3));
    vec3* z = (vec3*)malloc(size * sizeof(vec3));

    hipMalloc(&d_vectors, size * sizeof(Vector));
    hipMalloc(&d_z, size * sizeof(vec3));
    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int i = 0; i < size; i++) {
        z[i] = vec3();
    }

    MyKernel <<<blocksPerGrid, threadsPerBlock >> > (d_vectors, size, d_z);
    Vector h_vectors[size];
    hipMemcpy(h_vectors, d_vectors, size * sizeof(Vector), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, size * sizeof(vec3), hipMemcpyDeviceToHost);
    printVectors(h_vectors, size);
    for (int i = 0; i < size; i++) {
        cout << z[i] << endl;
    }
    hipFree(d_vectors);
    hipFree(d_z);



    cout << endl << endl << "Testowanie promieni" << WIDTH << HEIGHT;


    double focal_length = 10;
    point3 h_camera_center(5, 5, 5);
    point3 h_camera_focal(-5, -5, -5);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);
    dim3 threadsPerBlock2(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (HEIGHT + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    Generate_rays<<<numBlocks, threadsPerBlock2>>> (d_ray, focal_length, d_camera_center, d_camera_focal);
    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipFree(d_ray);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    free(h_ray[0]);
    free(h_ray);

    cout << endl << endl << "Bicie sciany" ;


    return 0;

}


