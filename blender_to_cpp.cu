
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");

    cout << "XDD" << endl;
    cout << endl << endl << " Linia przed cuda";


    const int size = 10;
    Vector* d_vectors;
    vec3* d_z = (vec3*)malloc(size * sizeof(vec3));
    vec3* z = (vec3*)malloc(size * sizeof(vec3));

    hipMalloc(&d_vectors, size * sizeof(Vector));
    hipMalloc(&d_z, size * sizeof(vec3));
    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int i = 0; i < size; i++) {
        z[i] = vec3();
    }

    MyKernel <<<blocksPerGrid, threadsPerBlock >> > (d_vectors, size, d_z);
    Vector h_vectors[size];
    hipMemcpy(h_vectors, d_vectors, size * sizeof(Vector), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, size * sizeof(vec3), hipMemcpyDeviceToHost);
    printVectors(h_vectors, size);
    for (int i = 0; i < size; i++) {
        cout << z[i] << endl;
    }
    hipFree(d_vectors);
    hipFree(d_z);



    cout << endl << endl << "Testowanie promieni";


    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(WIDTH * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }

    hipMalloc(&d_ray, WIDTH*HEIGHT * sizeof(ray));


    dim3 threadsPerBlock2(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock2.x - 1) / threadsPerBlock2.x,(HEIGHT + threadsPerBlock2.y - 1) / threadsPerBlock2.y);

    double focal_length = 10;
    point3 h_camera_center = point3(5, 5, 5);
    point3 h_camera_focal = point3(-5, -5, -5);
    point3* d_camera_center;
    point3* d_camera_focal;

    hipMalloc((point3)&d_camera_center, sizeof(point3));
    hipMalloc((point3)&d_camera_focal, sizeof(point3));
    hipMemcpy(&d_camera_center, h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(&d_camera_focal, h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);


    Generate_rays<<<numBlocks, threadsPerBlock2 >>> (h_ray[0], focal_length, d_camera_center, d_camera_focal);

    hipMemcpy(h_ray, d_ray, HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
  
    hipFree(d_ray);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    

    for(int i = 0; i < WIDTH; i++)
    {
        for (int j = 0; j < HEIGHT; j++) 
        {
            cout << "  " << h_ray[j][i] << "  ";
        }
        cout << endl;
    }



    return 0;

}


