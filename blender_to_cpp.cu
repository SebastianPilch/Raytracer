#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
#include<cmath>

void saveAsBMP(ray** img, int width, int height, const std::string& filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        return;
    }

    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding required by BMP format

    // BMP header
    int filesize = 54 + (3 * width + paddingSize) * height;
    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };

    // Write header
    file.write(fileHeader, 54);

    // Write pixel data
    for (int i = height - 1; i >= 0; i--) {
        for (int j = 0; j < width; j++) {
            float focal_len = 5.0f;
            unsigned char color = static_cast<unsigned char>(focal_len / (sqrt(img[i][j].dir.x() * img[i][j].dir.x() + img[i][j].dir.y() * img[i][j].dir.y())) / 4); // Scale value from [0, 1] to [0, 255]
            file.put(color);
            file.put(color);
            file.put(color);
        }

        // Add padding
        for (int k = 0; k < paddingSize; k++) {
            file.put(0);
        }
    }

    file.close();
}


void saveAsBMP2(float* angles, int width, int height, const std::string& filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        return;
    }

    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding required by BMP format

    // BMP header
    int filesize = 54 + (3 * width + paddingSize) * height;
    char fileHeader[54] = {
        'B', 'M',                         // Signature
        static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), // File size
        0,0, 0,0,                         // Reserved
        54,0,0,0,                         // File offset to pixel array
        40,0,0,0,                         // DIB header size
        static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), // Width
        static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), // Height
        1,0,                              // Planes
        24,0,                             // Bits per pixel
        0,0,0,0,                          // Compression
        0,0,0,0,                          // Image size (can be 0 for uncompressed)
        0,0,0,0,                          // X pixels per meter (unused)
        0,0,0,0,                          // Y pixels per meter (unused)
        0,0,0,0,                          // Total colors (0 means default)
        0,0,0,0                           // Important colors (0 means all are important)
    };

    // Write header
    file.write(fileHeader, 54);

    // Write pixel data
    for (int i = height - 1; i >= 0; i--) {
        for (int j = 0; j < width; j++) {
            // Retrieve RGB values from angles array
            float red = angles[(i * width + j) * 3 + 0];
            float green = angles[(i * width + j) * 3 + 1];
            float blue = angles[(i * width + j) * 3 + 2];

            unsigned char r = static_cast<unsigned char>(red * 255.0f);
            unsigned char g = static_cast<unsigned char>(green * 255.0f);
            unsigned char b = static_cast<unsigned char>(blue * 255.0f);

            file.put(b); // Blue channel
            file.put(g); // Green channel
            file.put(r); // Red channel
        }

        // Add padding
        for (int k = 0; k < paddingSize; k++) {
            file.put(0);
        }
    }

    file.close();
}


int main() {
    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1, face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2, face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");

    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");

    cout << endl << endl << "Kostka" << endl << endl;
    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;
    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    cout << endl << endl << "Testowanie promieni";

    // wybór obiektu
    int Vert_NUM = vert_num1;
    int Face_NUM = face_num1;
    int Normal_NUM = normal_num1;
    Pointer_storage liczony_objekt = Trojkatna_Kostka;
    // // // //

    float** Planes = new float* [Face_NUM];
    Planes[0] = new float[Face_NUM * 4];

    float** Verticies = new float* [Vert_NUM];
    Verticies[0] = new float[Vert_NUM * 3 * 4];

    float** Normals = new float* [Normal_NUM];
    Normals[0] = new float[Normal_NUM * 3];

    float* Distances = new float[WIDTH * HEIGHT * Face_NUM];
    float* ClosestNormals = new float[WIDTH * HEIGHT * Face_NUM * 3];

    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;

    int* start_face_at_index = new int[Face_NUM];

    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) {
        Length_to_Allocate_Faces += number_of_vertices_in_one_face[i];
    }

    int** Faces = new int* [Face_NUM];
    Faces[0] = new int[Length_to_Allocate_Faces];

    int current_index = 0;
    for (int i = 1; i < Face_NUM + 1; i++) {
        Faces[i] = Faces[0] + current_index + number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i] = current_index;
        current_index += number_of_vertices_in_one_face[i - 1];
        Planes[i] = Planes[0] + i * 4;
    }

    for (int i = 1; i < Vert_NUM; i++) {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++) {
        Normals[i] = Normals[0] + i * 3;
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < 4; j++)
            Planes[i][j] = liczony_objekt.Planes[i][j];
    }
    for (int i = 0; i < Vert_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Verticies[i][j] = liczony_objekt.Vertices[i][j];
    }
    for (int i = 0; i < Normal_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Normals[i][j] = liczony_objekt.Normals[i][j];
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < liczony_objekt.Face_size[i]; j++) {
            Faces[i][j] = liczony_objekt.Faces[i][j];
        }
    }

    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;
    float* d_closest_normals;

    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    hipMalloc(&d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float));
    hipMalloc(&d_closest_normals, WIDTH * HEIGHT * Face_NUM * 3 * sizeof(float));

    hipMemcpy(d_Faces, Faces[0], Length_to_Allocate_Faces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Planes, Planes[0], 4 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Normals, Normals[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Vertices, Verticies[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_vertices_in_one_face, number_of_vertices_in_one_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_normal_index_to_face, normal_index_to_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_face_at_index, start_face_at_index, Face_NUM * sizeof(int), hipMemcpyHostToDevice);

    double focal_length = 10.0;
    point3 h_camera_center(0.0, 0.0, -25.0);
    point3 h_camera_focal(0.0, 0.0, 0.0);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);

    dim3 blockDim(8, 8, 8);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    Generate_rays << <gridDim, blockDim >> > (d_ray, focal_length, d_camera_center, d_camera_focal, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, d_closest_normals);

    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipMemcpy(Distances, d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Planes[0], d_Planes, 3 * Face_NUM * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(ClosestNormals, d_closest_normals, WIDTH * HEIGHT * Face_NUM * 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    hipFree(d_closest_normals);

    saveAsBMP(h_ray, WIDTH, HEIGHT, "result_image.bmp");
    saveAsBMP2(ClosestNormals, WIDTH, HEIGHT, "normals_image.bmp");

    free(h_ray[0]);
    free(h_ray);
    delete[] ClosestNormals;

    return 0;
}

