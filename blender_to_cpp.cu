
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");


    cout << endl << endl << "Testowanie promieni" ;












    //wybór obiektu
    int Vert_NUM = vert_num1;
    int Face_NUM = face_num1;
    int Normal_NUM = normal_num1;
    Pointer_storage liczony_objekt = Kostka;
    // // // //


    int** Faces = liczony_objekt.Faces;
    float** Verticies = liczony_objekt.Vertices;
    float** Normals = liczony_objekt.Normals;
    float** Planes = liczony_objekt.Planes;
    float* Distances = new float[WIDTH*HEIGHT*Face_NUM];
    // (face_idx * WIDTH * HEIGHT) + (h * WIDTH) + w;


    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;

    int* start_face_at_index = new int[Face_NUM];

    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) { Length_to_Allocate_Faces += number_of_vertices_in_one_face[i]; }

    int* Faces_d = new int[Length_to_Allocate_Faces];
    float* Vertices_d, Normals_d, Planes_d;
    int current_index = 0;
    for (int i = 1; i < Face_NUM; i++)
    {
        Faces[i] = Faces[current_index] + number_of_vertices_in_one_face[i - 1];
        current_index += number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i] = current_index;
        Planes[i] = Planes[0] + i * 4;

    }
    for (int i = 1; i < Vert_NUM; i++)
    {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++)
    {
        Normals[i] = Normals[0] + i * 3;
    }
    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;


    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    hipMalloc(&d_distances, WIDTH*HEIGHT*Face_NUM * sizeof(float));



    double focal_length = 10;
    point3 h_camera_center(5, 5, 5);
    point3 h_camera_focal(-5, -5, -5);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);
    

    dim3 blockDim(512, 512, 512); 
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    Generate_rays<<< gridDim, blockDim >>> (d_ray, focal_length, d_camera_center, d_camera_focal, normal_index_to_face,number_of_vertices_in_one_face,
    Faces[0], Verticies[0], Normals[0], Planes[0], d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances);


    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipMemcpy(Distances, d_distances, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);

    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);


    cout << endl << endl << "Bicie sciany" ;

    for (int i = 0; i < WIDTH; i++) 
    {
        for (int j = 0; j < HEIGHT; j++) 
        {
            for (int f = 0; f < Face_NUM; f++) 
            {
                if (Distances[(f * WIDTH * HEIGHT) + (j * WIDTH) + i] > 0)
                {
                    cout << "  " << Distances[(f * WIDTH * HEIGHT) + (j * WIDTH) + i] << "  ";
                }
            }
        
        }

    
    }




    free(h_ray[0]);
    free(h_ray);

    cout << endl<<Length_to_Allocate_Faces;























    return 0;

}


