
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");


    cout << endl << endl << "Testowanie promieni" ;












    //wybór obiektu
    int Vert_NUM = vert_num3;
    int Face_NUM = face_num3;
    int Normal_NUM = normal_num3;
    Pointer_storage liczony_objekt = pociety_walec;
    // // // //


    int** Faces = liczony_objekt.Faces;
    float** Verticies = liczony_objekt.Vertices;
    float** Normals = liczony_objekt.Normals;
    float** Planes = liczony_objekt.Planes;
    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;

    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) { Length_to_Allocate_Faces += number_of_vertices_in_one_face[i]; }

    int* Faces_d = new int[Length_to_Allocate_Faces];
    float* Vertices_d, Normals_d, Planes_d;
    int current_index = 0;
    for (int i = 1; i < Face_NUM; i++)
    {
        Faces[i] = Faces[current_index] + number_of_vertices_in_one_face[i - 1];
        current_index += number_of_vertices_in_one_face[i - 1];
        Planes[i] = Planes[0] + i * 4;

    }
    for (int i = 1; i < Vert_NUM; i++)
    {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++)
    {
        Normals[i] = Normals[0] + i * 3;
    }
    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;


    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));



    double focal_length = 10;
    point3 h_camera_center(5, 5, 5);
    point3 h_camera_focal(-5, -5, -5);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);
    dim3 threadsPerBlock2(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock2.x - 1) / threadsPerBlock2.x, (HEIGHT + threadsPerBlock2.y - 1) / threadsPerBlock2.y);
    Generate_rays<<<numBlocks, threadsPerBlock2>>> (d_ray, focal_length, d_camera_center, d_camera_focal, normal_index_to_face,number_of_vertices_in_one_face,
     Faces[0], Verticies[0], Normals[0], Planes[0]);
    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipFree(d_ray);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    free(h_ray[0]);
    free(h_ray);

    cout << endl << endl << "Bicie sciany" ;






    cout << endl<<Length_to_Allocate_Faces;























    return 0;

}


