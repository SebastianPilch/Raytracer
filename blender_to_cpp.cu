#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
#include "Material.cuh"
#include "SaveAsBMP.cuh"
#include<cmath>


int main() {

    ///////////////////////////////////////////////
    //
    //wczytanie obiektów .obj
    //
    //////////////////////////////////////////////////


    //int vert_num1 = 3;
    //int face_num1 = 1;
    //int normal_num1 = 1;

    //Pointer_storage Kostka = GetDataFromObj(vert_num1, face_num1, normal_num1, "../../../helpers/untitled.obj");

    //int vert_num2 = 3;
    //int face_num2 = 1;
    //int normal_num2 = 1;

    //Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2, face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");
    int object_couter = 0;
    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    //Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");
    Pointer_storage pociety_walec = GetDataFromObj(vert_num3, face_num3, normal_num3, object_couter, "../../../helpers/scena_jeszce_raz.obj");

    //cout << endl << endl << "Kostka" << endl << endl;

    //cout << endl << endl << "Kostka" << endl << endl;
    //cout << endl << endl << "Kostka" << endl << endl;






    //Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    //cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    //Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    //cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;
    //Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //cout << endl << endl << "Testowanie promieni";



    ///////////////////////////////////////////////
    //
    //wybór obiektu
    //
    //////////////////////////////////////////////////
    int Vert_NUM = vert_num3;
    int Face_NUM = face_num3;
    int Normal_NUM = normal_num3;


    cout << object_couter << endl;

    Pointer_storage liczony_objekt = pociety_walec;
    ///////////////////////////////////////////////
    //
    //przepisanie wskaźników
    //
    //////////////////////////////////////////////////
    float** Planes = new float* [Face_NUM];
    Planes[0] = new float[Face_NUM * 4];

    float** Verticies = new float* [Vert_NUM];
    Verticies[0] = new float[Vert_NUM * 3];

    float** Normals = new float* [Normal_NUM];
    Normals[0] = new float[Normal_NUM * 3];

    float* Distances = new float[WIDTH * HEIGHT * Face_NUM];
    float* Colors = new float[WIDTH * HEIGHT * 3];
    float* shadows = new float[WIDTH * HEIGHT * 3];

    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;
    int* Object_to_Face = liczony_objekt.Object_to_Face;
    int* Object_to_Vertex = liczony_objekt.Object_to_Vertex;
    int* start_face_at_index = new int[Face_NUM];
    float* Intersections = new float[WIDTH * HEIGHT * 3];
    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) {
        Length_to_Allocate_Faces += number_of_vertices_in_one_face[i];
    }

    int** Faces = new int* [Face_NUM];
    Faces[0] = new int[Length_to_Allocate_Faces];

    int current_index = 0;
    for (int i = 1; i < Face_NUM + 1; i++) {
        Faces[i] = Faces[0] + current_index + number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i - 1] = current_index;
        current_index += number_of_vertices_in_one_face[i - 1];
        Planes[i] = Planes[0] + i * 4;


        //cout << i-1 << "  :  " << start_face_at_index[i-1] << endl;
    }

    for (int i = 1; i < Vert_NUM; i++) {
        Verticies[i] = Verticies[0] + i * 3;
    }
    for (int i = 1; i < Normal_NUM; i++) {
        Normals[i] = Normals[0] + i * 3;
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < 4; j++)
        {
            Planes[i][j] = liczony_objekt.Planes[i][j];
        }
    }
    for (int i = 0; i < Vert_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Verticies[i][j] = liczony_objekt.Vertices[i][j];
    }
    for (int i = 0; i < Normal_NUM; i++) {
        for (int j = 0; j < 3; j++)
            Normals[i][j] = liczony_objekt.Normals[i][j];
    }
    for (int i = 0; i < Face_NUM; i++) {
        for (int j = 0; j < liczony_objekt.Face_size[i]; j++) {
            Faces[i][j] = liczony_objekt.Faces[i][j];
        }
    }

    ///////////////////////////////////////////////
    //
    //utworzenie materiałów
    //
    //////////////////////////////////////////////////
    Material* Materials = new Material[4];
    Materials[0] = Material(0.8f, 0.1f, 0.1f,   // diffuse (red)
        0.9f, 0.6f, 0.6f,   // specular (light red)
        0.3f, 0.1f, 0.1f,   // ambient (dark red)
        1.0f,               // alpha
        4.0f,  	        // shininess  
        0.5f);             // reflectivity

    Materials[1] = Material(0.1f, 0.8f, 0.1f,   // diffuse (green)
        0.6f, 0.9f, 0.6f,   // specular (light green)
        0.1f, 0.3f, 0.1f,   // ambient (dark green)
        1.0f,               // alpha
        4.0f,  	        // shininess  
        0.5f);             // reflectivity

    Materials[2] = Material(0.1f, 0.1f, 0.8f,   // diffuse (blue)
        0.9f, 0.9f, 0.9f,   // specular (light blue)
        0.1f, 0.1f, 0.3f,   // ambient (dark blue)
        1.0f,               // alpha
        4.0f,  	        // shininess  
        0.5f);             // reflectivity

    Materials[3] = Material(0.8f, 0.8f, 0.1f,   // diffuse (yellow)
        0.9f, 0.9f, 0.6f,   // specular (light yellow)
        0.3f, 0.3f, 0.1f,   // ambient (dark yellow)
        1.0f,               // alpha
        4.0f,  	        // shininess  
        0.5f);             // reflectivity

    ///////////////////////////////////////////////
    //
    //Alokacja i przekazanie danych do karty
    //
    //
    //
    //
    //////////////////////////////////////////////////

    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    int* d_Object_to_Vertex;
    int* d_Object_to_Face;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;
    float* d_closest_interesections;
    float* d_shadows;
    Material* d_Materials;

    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    hipMalloc(&d_distances, WIDTH * HEIGHT * Face_NUM * sizeof(float));
    hipMalloc(&d_closest_interesections, WIDTH * HEIGHT * 3 * sizeof(float));
    hipMalloc(&d_Object_to_Vertex, Vert_NUM * sizeof(int));
    hipMalloc(&d_Object_to_Face, Face_NUM * sizeof(int));
    hipMalloc(&d_shadows, WIDTH* HEIGHT * 3 * sizeof(float));


    hipMemcpy(d_Faces, Faces[0], Length_to_Allocate_Faces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Planes, Planes[0], 4 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Normals, Normals[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Vertices, Verticies[0], 3 * Vert_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_vertices_in_one_face, number_of_vertices_in_one_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_normal_index_to_face, normal_index_to_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_face_at_index, start_face_at_index, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Object_to_Vertex, Object_to_Vertex, Vert_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Object_to_Face, Object_to_Face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);

    ///////////////////////////////////////////////
    //
    //Tranformacja obiektu rotacja/skala/przesunięcie
    //
    //////////////////////////////////////////////////

    int threadsPerBlock = 256;
    int blocksPerGrid = (Vert_NUM + threadsPerBlock - 1) / threadsPerBlock;
    float TranslateX = 0.0f;
    float TranslateY = 0.0f;
    float TranslateZ = 0.0f;
    float rotateX = 0.0f;
    float rotateY = 0.0f;
    float rotateZ = 0.0f;
    float scaleX = 1.0f;
    float scaleY = -1.0f;
    float scaleZ = -1.0f;

    int index = 0;
    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 1;
    TranslateZ = 5.0f;
    TranslateY = 0.3f;


    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 2;
    TranslateZ = 0.0f;

    Transform <<< blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();
    index = 3;
    TranslateY = -1.0f;

    Transform << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, Vert_NUM, d_Object_to_Vertex, index, TranslateX, TranslateY, TranslateZ, rotateX, rotateY, rotateZ, scaleX, scaleY, scaleZ);
    hipDeviceSynchronize();

    threadsPerBlock = 256;
    blocksPerGrid = (Face_NUM + threadsPerBlock - 1) / threadsPerBlock;
    Update_normals_and_Planes << <blocksPerGrid, threadsPerBlock >> > (d_Vertices, d_Faces, d_Object_to_Face, index, d_Normals, d_Planes, d_number_of_vertices_in_one_face, d_normal_index_to_face, d_start_face_at_index, Face_NUM, Normal_NUM);
    hipDeviceSynchronize();

    ///////////////////////////////////////////////
    //
    //  Wyznaczanie Promieni, uderzenia i dystanse
    //
    //////////////////////////////////////////////////
    int reflecions = 0;
    double focal_length = 10;
    point3 h_camera_center(120.0, -60.0, -50.0);
    point3  h_camera_focal(60.0 / 2, -25.0 / 2, -25.0 / 2);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);



    dim3 blockDim(8, 8, 8);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    ///////////////////////////////////////////////
    //
    // wyznaczenie promieni
    //
    ///////////////////////////////////////////////
    hipMemcpy(d_ray, h_ray[0], WIDTH * HEIGHT * sizeof(ray), hipMemcpyHostToDevice);


    Generate_rays <<< gridDim, blockDim >>> (d_ray, focal_length, d_camera_center, d_camera_focal);
    hipDeviceSynchronize();

    ///////////////////////////////////////////////
    //
    // wyznaczenie wektora dystansów
    //
    ///////////////////////////////////////////////

    Generate_distances << <gridDim, blockDim >> > (d_ray, d_camera_center, d_closest_interesections, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, reflecions);
    hipDeviceSynchronize();
    reflecions += 1;




    ///////////////////////////////////////////////
    //
    // dobór najbliższej ściany z wektora dystansów
    //
    ////////////////////////////////////////////////


    int* d_close_indexes;
    float* d_colors;

    hipMalloc(&d_Materials, object_couter * sizeof(Material));
    hipMalloc(&d_close_indexes, WIDTH * HEIGHT * sizeof(int));
    hipMalloc(&d_colors, WIDTH * HEIGHT * 3 * sizeof(float));


    hipMemcpy(d_Materials, Materials, object_couter * sizeof(Material), hipMemcpyHostToDevice);

    Choose_closest <<< gridDim, blockDim >>> (d_distances, Face_NUM, d_colors, d_Planes, d_Materials, d_ray, d_Object_to_Face, d_close_indexes, d_closest_interesections);
    hipDeviceSynchronize();

    hipMemcpy(Colors, d_colors, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);

    saveAsBMP(Colors, WIDTH, HEIGHT, "normals_image.bmp");
    ///////////////////////////////////////////////
    //
    // Dodanie cienia
    //
    ////////////////////////////////////////////////


    Add_shadows <<< gridDim, blockDim >> > (d_closest_interesections, d_shadows, d_normal_index_to_face, d_number_of_vertices_in_one_face, d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM);
    hipDeviceSynchronize();

    hipMemcpy(shadows, d_shadows, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);


    saveAsBMP(shadows, WIDTH, HEIGHT, "shadowes_image.bmp");

    for (int i = 0; i < WIDTH * HEIGHT * 3; i++)
    {
        if (shadows[i] >= 1)
        {
            Colors[i] -= 0.1f;
            if (Colors[i] < 0) { Colors[i] = 0.0f; }
        }

    }
    saveAsBMP(Colors, WIDTH, HEIGHT, "shadowed_scene_image.bmp");


    ///////////////////////////////////////////////
    //
    // druga iteracja promieni
    //
    ////////////////////////////////////////////////
    Update_rays << < gridDim, blockDim >>> (d_ray, d_closest_interesections, d_close_indexes, d_normal_index_to_face, d_Normals);
    hipDeviceSynchronize();


    Generate_distances << < gridDim, blockDim >> > (d_ray, d_camera_center, d_closest_interesections, d_normal_index_to_face, d_number_of_vertices_in_one_face,
        d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances, reflecions);
    hipDeviceSynchronize();
    reflecions += 1;


    hipMemcpy(d_Materials, Materials, object_couter * sizeof(Material), hipMemcpyHostToDevice);

    Choose_closest << <gridDim, blockDim >> > (d_distances, Face_NUM, d_colors, d_Planes, d_Materials, d_ray, d_Object_to_Face, d_close_indexes, d_closest_interesections);
    hipDeviceSynchronize();


    hipMemcpy(Colors, d_colors, WIDTH * HEIGHT * 3 * sizeof(float), hipMemcpyDeviceToHost);
    saveAsBMP(Colors, WIDTH, HEIGHT, "reflected_image.bmp");


    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);
    hipFree(d_closest_interesections);

    free(h_ray[0]);
    free(h_ray);
    delete[] Colors;

    return 0;
}