
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//
//void saveAsBMP(const std::vector<std::vector<float>>& img, int width, int height, const std::string& filename) {
//    std::ofstream file(filename, std::ios::out | std::ios::binary);
//
//    if (!file) {
//        std::cerr << "Cannot open file: " << filename << std::endl;
//        return;
//    }
//
//    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding wymagany przez format BMP
//
//    // Nagłówek BMP
//    int filesize = 54 + (3 * width + paddingSize) * height;
//    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };
//
//    // Zapisanie nagłówka
//    file.write(fileHeader, 54);
//
//    // Zapisanie danych pikseli
//    for (int i = height - 1; i >= 0; i--) {
//        for (int j = 0; j < width; j++) {
//            unsigned char color = static_cast<unsigned char>(img[i][j] * 255); // Skalowanie wartości z [0, 1] do [0, 255]
//            file.put(color);
//            file.put(color);
//            file.put(color);
//        }
//        // Dodanie paddingu
//        for (int k = 0; k < paddingSize; k++) {
//            file.put(0);
//        }
//    }
//
//    file.close();
//}



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");

    cout << "XDD" << endl;
    cout << endl << endl << " Linia przed cuda";
    const int size = 10;
    Vector* d_vectors;
    vec3* d_z = (vec3*)malloc(size * sizeof(vec3));
    vec3* z = (vec3*)malloc(size * sizeof(vec3));

    hipMalloc(&d_vectors, size * sizeof(Vector));
    hipMalloc(&d_z, size * sizeof(vec3));
    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    for (int i = 0; i < size; i++) {
        z[i] = vec3();
    }

    MyKernel <<<blocksPerGrid, threadsPerBlock >> > (d_vectors, size, d_z);
    Vector h_vectors[size];
    hipMemcpy(h_vectors, d_vectors, size * sizeof(Vector), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, size * sizeof(vec3), hipMemcpyDeviceToHost);
    printVectors(h_vectors, size);
    for (int i = 0; i < size; i++) {
        cout << z[i] << endl;
    }
    hipFree(d_vectors);
    hipFree(d_z);

    return 0;

}


