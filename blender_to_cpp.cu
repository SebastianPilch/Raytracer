
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"
//



int main() {


    int vert_num1 = 3;
    int face_num1 = 1;
    int normal_num1 = 1;

    Pointer_storage Kostka = GetDataFromObj(vert_num1,
        face_num1, normal_num1, "../../../helpers/untitled.obj");

    int vert_num2 = 3;
    int face_num2 = 1;
    int normal_num2 = 1;

    Pointer_storage Trojkatna_Kostka = GetDataFromObj(vert_num2,
        face_num2, normal_num2, "../../../helpers/Trojkatny_szescian.obj");



    int vert_num3 = 3;
    int face_num3 = 1;
    int normal_num3 = 1;

    Pointer_storage pociety_walec = GetDataFromObj(vert_num3,
        face_num3, normal_num3, "../../../helpers/walec_ale_kanciastyXD.obj");


    cout << endl << endl << "Kostka" << endl << endl;

    Print_Import_data(Kostka, vert_num1, normal_num1, face_num1);

    cout << endl << endl << "Sześcian pokrojony" << endl << endl;
    Print_Import_data(Trojkatna_Kostka, vert_num2, normal_num2, face_num2);

    cout << endl << endl << "zlosliwy przyklad kanciastego walca" << endl << endl;

    Print_Import_data(pociety_walec, vert_num3, normal_num3, face_num3);

    //saveAsBMP(img, width, height, "result_image.bmp");


    cout << endl << endl << "Testowanie promieni" ;

    //wybór obiektu
    int Vert_NUM = vert_num1;
    int Face_NUM = face_num1;
    int Normal_NUM = normal_num1;
    Pointer_storage liczony_objekt = Kostka;
    // // // //

    float** Planes = new float*[Face_NUM];
    Planes[0] = new float[Face_NUM * 4];
  
    int** Faces = new int* [Face_NUM];
    Faces[0] = new int[Face_NUM * 3];

    float** Verticies = new float* [Vert_NUM];
    Verticies[0] = new float[Vert_NUM * 3 * 4];

    float** Normals = new float*[Normal_NUM];
    Normals[0] = new float[Normal_NUM * 3];

    float* Distances = new float[WIDTH*HEIGHT*Face_NUM];
    // (face_idx * WIDTH * HEIGHT) + (h * WIDTH) + w;

    int* number_of_vertices_in_one_face = liczony_objekt.Face_size;
    int* normal_index_to_face = liczony_objekt.Face_to_Normal;

    int* start_face_at_index = new int[Face_NUM];

    start_face_at_index[0] = 0;
    int Length_to_Allocate_Faces = 0;
    for (int i = 0; i < Face_NUM; i++) { Length_to_Allocate_Faces += number_of_vertices_in_one_face[i]; }

    int current_index = 0;
    for (int i = 1; i < Face_NUM; i++)
    {
        Faces[i] = Faces[current_index] + number_of_vertices_in_one_face[i - 1];
        current_index += number_of_vertices_in_one_face[i - 1];
        start_face_at_index[i] = current_index;
        Planes[i] = Planes[0] + i * 4;

    }
    Planes = liczony_objekt.Planes;
    for (int i = 0; i < Face_NUM; i++) 
    {
        for (int j = 0; j < 4; j++)
        Planes[i][j] = liczony_objekt.Planes[i][j];
    }

    for (int i = 0; i < Face_NUM*4; i++)
    {
            std::cout << "  " << Planes[i] << "  ";
        std::cout << endl;
    }

    Faces = liczony_objekt.Faces;
    for (int i = 1; i < Vert_NUM; i++)
    {
        Verticies[i] = Verticies[0] + i * 3;
    }

    Verticies = liczony_objekt.Vertices;

    for (int i = 1; i < Normal_NUM; i++)
    {
        Normals[i] = Normals[0] + i * 3;
    }

    Normals = liczony_objekt.Normals;
    int* d_Faces;
    int* d_number_of_vertices_in_one_face;
    int* d_normal_index_to_face;
    int* d_start_face_at_index;
    float* d_distances;
    float* d_Vertices;
    float* d_Normals;
    float* d_Planes;


    hipMalloc(&d_Faces, Length_to_Allocate_Faces * sizeof(int));
    hipMalloc(&d_Planes, 4 * Face_NUM * sizeof(float));
    hipMalloc(&d_Normals, 3 * Normal_NUM * sizeof(float));
    hipMalloc(&d_Vertices, 3 * Vert_NUM * sizeof(float));
    hipMalloc(&d_number_of_vertices_in_one_face, Face_NUM * sizeof(int));
    hipMalloc(&d_normal_index_to_face, Face_NUM * sizeof(int));
    hipMalloc(&d_start_face_at_index, Face_NUM * sizeof(int));
    
    
    
    hipMalloc(&d_distances, WIDTH*HEIGHT*Face_NUM * sizeof(float));


    hipMemcpy(d_Faces, Faces[0], Length_to_Allocate_Faces * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Planes, Planes[0], 4 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Normals, Normals[0], 3 * Face_NUM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Vertices, Verticies[0], 3 * Face_NUM *  sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_number_of_vertices_in_one_face, &number_of_vertices_in_one_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_normal_index_to_face, &normal_index_to_face, Face_NUM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_start_face_at_index, &start_face_at_index, Face_NUM * sizeof(int), hipMemcpyHostToDevice);

    double focal_length = 10;
    point3 h_camera_center(5, 5, 5);
    point3 h_camera_focal(-5, -5, -5);
    point3* d_camera_center;
    point3* d_camera_focal;
    ray** h_ray;
    ray* d_ray;
    h_ray = (ray**)malloc(HEIGHT * sizeof(ray*));
    h_ray[0] = (ray*)malloc(HEIGHT * WIDTH * sizeof(ray));
    for (int i = 1; i < HEIGHT; i++) {
        h_ray[i] = h_ray[0] + i * WIDTH;
    }
    hipMalloc(&d_ray, WIDTH * HEIGHT * sizeof(ray));
    hipMalloc((void**)&d_camera_center, sizeof(point3));
    hipMalloc((void**)&d_camera_focal, sizeof(point3));
    hipMemcpy(d_camera_center, &h_camera_center, sizeof(point3), hipMemcpyHostToDevice);
    hipMemcpy(d_camera_focal, &h_camera_focal, sizeof(point3), hipMemcpyHostToDevice);
    

    dim3 blockDim(8, 8, 8); 
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x,
        (HEIGHT + blockDim.y - 1) / blockDim.y,
        (Face_NUM + blockDim.z - 1) / blockDim.z);

    Generate_rays<<< gridDim, blockDim >>> (d_ray, focal_length, d_camera_center, d_camera_focal, d_normal_index_to_face,d_number_of_vertices_in_one_face,
    d_Faces, d_Vertices, d_Normals, d_Planes, d_start_face_at_index, Face_NUM, Vert_NUM, Normal_NUM, d_distances);


    hipMemcpy(h_ray[0], d_ray, WIDTH * HEIGHT * sizeof(ray), hipMemcpyDeviceToHost);
    hipMemcpy(Distances, d_distances, WIDTH * HEIGHT*Face_NUM * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Planes[0], d_Planes,3 * Face_NUM * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_ray);
    hipFree(d_distances);
    hipFree(d_camera_center);
    hipFree(d_camera_focal);

    //cout << endl << endl << "Bicie sciany" ;
    //for (int i = 0; i < WIDTH; i++)
    //{
    //    for (int j = 0; j < HEIGHT; j++)
    //    {
    //        if(h_ray[i][j].orig[0] == 5)
    //        {
    //            cout << "  " << h_ray[i][j] << "  ";
    //        }
    //    }
    //}

    //for (int i = 0; i < WIDTH; i++) 
    //{
    //    for (int j = 0; j < HEIGHT; j++) 
    //    {
    //        for (int f = 0; f < Face_NUM; f++) 
    //        {
    //              cout << "  " << Distances[(j * WIDTH * Face_NUM) + (i * Face_NUM) + f] << "  ";
    //        }
    //    }
    //}




    free(h_ray[0]);
    free(h_ray);

    cout << endl<<Length_to_Allocate_Faces;

    return 0;

}


