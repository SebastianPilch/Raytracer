#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <iomanip>
#include "vec3.cuh"
#include "Parllel_fun.cuh"
#include "Ray.cuh"
using namespace std;
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#define WIDTH 400
#define HEIGHT 400
#define VIEWPORT_HEIGHT  2.0
#define VIEWPORT_WIDTH VIEWPORT_HEIGHT * (double)(WIDTH / HEIGHT)
#define VIEWPORT_U vec3(VIEWPORT_WIDTH, 0, 0)
#define VIEWPORT_V vec3(0, -VIEWPORT_HEIGHT, 0)
#define DELTA_U VIEWPORT_U / WIDTH
#define DELTA_V VIEWPORT_V / HEIGHT

//__global__ void Matrixadd(int m, int n, float* d_x, float* d_y, float* d_z)
//{
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i < m * n)
//    {
//        d_z[i] = d_x[i] + d_y[i];
//    }
//
//}


//void MatrixAddition(int N, int M, float** x, float** y, float** z)
//{
//    x[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) x[i] = x[0] + i * N;
//    y[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) y[i] = y[0] + i * N;
//    z[0] = new float[M * N];
//    for (size_t i = 1; i < M; i++) z[i] = z[0] + i * N;
//    float* d_x, *d_y, *d_z;
//    hipMalloc(&d_x, N * M * sizeof(float));
//    hipMalloc(&d_y, N * M * sizeof(float));
//    hipMalloc(&d_z, N * M * sizeof(float));
//    hipMemcpy(d_x, x[0], N * M * sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(d_y, y[0], N * M * sizeof(float), hipMemcpyHostToDevice);
//    Matrixadd << <M, N >> > (M, N, d_x, d_y, d_z);
//    hipMemcpy(z[0], d_z, N * M * sizeof(float), hipMemcpyDeviceToHost);
//    hipFree(d_x);
//    hipFree(d_y);
//    hipFree(d_z);
//    delete[] x[0];
//    delete[] x;
//    delete[] y[0];
//    delete[] y;
//    delete[] z[0];
//    delete[] z;
//}

//int img_height = 400;
//int img_width = 400;
//
//auto focal_length = 1.0;
//auto viewport_height = 2.0;
//auto viewport_width = viewport_height * (double(img_width) / img_height);
//auto camera_center = point3(0, 0, 5);
//auto viewport_u = vec3(viewport_width, 0, 0);
//auto viewport_v = vec3(0, -viewport_height, 0);
//
//auto pixel_delta_u = viewport_u / img_width;
//auto pixel_delta_v = viewport_v / img_height;
//auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
//auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);
//
//
//std::vector<std::vector<float>> img(img_height, std::vector<float>(img_width));
//
//for (int j = 0; j < img_height; j++) {
//    for (int i = 0; i < img_width; i++) {
//        auto pixel_center = pixel00_loc + (i * pixel_delta_u) + (j * pixel_delta_v);
//        auto ray_direction = pixel_center - camera_center;
//        ray newRay = ray(camera_center, ray_direction);
//
//        float hit_anything = 0.0;
//        for (const auto& pair : Planes_to_faces) {
//            bool hit = Face_hit(pair.second, newRay, vertices_to_faces[pair.first], vertices_coors);
//            if (hit) {
//                hit_anything += 0.2;
//            }
//
//        }
//
//        img[j][i] = hit_anything;
//    }
//}



//__global__ void Image_matrix(int** img,double focal_length, point3 camera_center, point3 camera_focal)
//{
//    int i = threadIdx.x + blockIdx.x * blockDim.x;
//    int j = threadIdx.y + blockIdx.y * blockDim.y;

    //auto viewport_upper_left = camera_center - vec3(0, 0, focal_length) - VIEWPORT_U / 2 - VIEWPORT_V / 2;
    //auto pixel00_loc = viewport_upper_left + 0.5 * (DELTA_U + DELTA_V);

    //auto pixel_center = pixel00_loc + (i * DELTA_U) + (j * DELTA_V);
    //auto ray_direction = pixel_center - camera_center;
    //ray newRay = ray(camera_center, ray_direction);
    //float hit_anything = 0.0;

    //bool hit = Face_hit(pair.second, newRay, vertices_to_faces[pair.first], vertices_coors);
    //if (hit) 
    //{
    //   hit_anything += 0.2;
    //}
    //img[j][i] = hit_anything;

//}
//dim3 Image_size(width, height);
