﻿// blender_to_cpp.cpp: definiuje punkt wejścia dla aplikacji.


#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include "vec3.cuh"
#include "ImportObj.cuh"
#include "Ray.cuh"
#include "Parllel_fun.cuh"

using namespace std;

void saveAsBMP(const std::vector<std::vector<float>>& img, int width, int height, const std::string& filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);

    if (!file) {
        std::cerr << "Cannot open file: " << filename << std::endl;
        return;
    }

    int paddingSize = (4 - (width * 3) % 4) % 4; // Padding wymagany przez format BMP

    // Nagłówek BMP
    int filesize = 54 + (3 * width + paddingSize) * height;
    char fileHeader[54] = { 'B', 'M', 0,0,0,0, 0,0, 0,0, 54,0,0,0, 40,0,0,0, static_cast<char>(width), static_cast<char>(width >> 8), static_cast<char>(width >> 16), static_cast<char>(width >> 24), static_cast<char>(height), static_cast<char>(height >> 8), static_cast<char>(height >> 16), static_cast<char>(height >> 24), 1,0, 24,0, 0,0,0,0, static_cast<char>(filesize), static_cast<char>(filesize >> 8), static_cast<char>(filesize >> 16), static_cast<char>(filesize >> 24), 0,0,0,0, 0,0,0,0 };

    // Zapisanie nagłówka
    file.write(fileHeader, 54);

    // Zapisanie danych pikseli
    for (int i = height - 1; i >= 0; i--) {
        for (int j = 0; j < width; j++) {
            unsigned char color = static_cast<unsigned char>(img[i][j] * 255); // Skalowanie wartości z [0, 1] do [0, 255]
            file.put(color);
            file.put(color);
            file.put(color);
        }
        // Dodanie paddingu
        for (int k = 0; k < paddingSize; k++) {
            file.put(0);
        }
    }

    file.close();
}



int main() {

    int vert_num = 3;
    float** vert = new float* [vert_num];

    for (int i = 0; i < vert_num; ++i) {
        vert[i] = new float[3];  // Allocating 3 floats for each row
    }

    map<int, vector<int>> vertices_to_faces;
    map<int, vec3> faces_normals;
    map<int, Plane> Planes_to_faces;
    GetDataFromObj(vert, vert_num,  vertices_to_faces, faces_normals,Planes_to_faces, "..\\..\\..\\helpers\\Trojkatny_szescian.obj");



    //saveAsBMP(img, width, height, "result_image.bmp");





    //float** x = new float* [4];
    //float** y = new float* [4];
    //float** z= new float* [4];



    //MatrixAddition(4,4,x,y,z);


    //for (int i = 0; i < 4; i++) {
    //    for (int j = 0; j < 4; j++) 
    //    {
    //        cout <<" " << z[i][j];

    //    }
    //    cout << endl;
    //}












    return 0;
}