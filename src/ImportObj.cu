#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "ImportObj.cuh"
#include <hip/hip_runtime.h>
#include <>

using namespace std;

size_t split(const std::string& txt, std::vector<std::string>& strs, char ch)
{
    size_t pos = txt.find(ch);
    size_t initialPos = 0;
    strs.clear();

    // Decompose statement
    while (pos != std::string::npos) {
        strs.push_back(txt.substr(initialPos, pos - initialPos));
        initialPos = pos + 1;

        pos = txt.find(ch, initialPos);
    }

    // Add the last one
    strs.push_back(txt.substr(initialPos, std::min(pos, txt.size()) - initialPos + 1));

    return strs.size();
}


void GetDataFromObj(float*& vertices_coors, int& Vertices_coords_size, map<int, vector<int>>& vertices_to_faces, map<int, vec3>& faces_normals, map<int, Plane>& Planes_to_faces, string file_path)
{
    ifstream plik;
    plik.open(file_path);
    if (!plik.is_open()) {
        cerr << "Nie uda�o si� otworzy� pliku: " << file_path << endl;
        return;
    }
    string linia;

    vector<string> Splited_line;

    int index_v = 0;
    int index_f = 0;
    int index_normal = 0;
    bool UV_map = 0;


    while (getline(plik, linia)) {
        split(linia, Splited_line, ' ');
        if (Splited_line[0] == "v") 
        {
            if (index_v >= Vertices_coords_size) 
            {
                size_t newSize = Vertices_coords_size * 2;
                float** newVertices = new float* [newSize];
                for (size_t i = 0; i < newSize; ++i) 
                {
                    newVertices[i] = new float[3];
                }
                for (size_t i = 0; i < index_v; ++i) 
                {
                    newVertices[i][0] = vertices_coors[i][0];
                    newVertices[i][1] = vertices_coors[i][1];
                    newVertices[i][2] = vertices_coors[i][2];
                    delete[] vertices_coors[i]; // Usuni�cie starej pami�ci
                }
                delete[] vertices_coors; // Usuni�cie starej tablicy wska�nik�w
                vertices_coors = newVertices;
                Vertices_coords_size = newSize;
            }

            vertices_coors[index_v * 3] = (float)stof(Splited_line[1]);
            vertices_coors[index_v * 3 + 1] = (float)stof(Splited_line[2]);
            vertices_coors[index_v * 3 + 2] = (float)stof(Splited_line[3]);
            index_v++;
        }
    




            if (Splited_line[0] == "vn")
            {

            }
            if (Splited_line[0] == "f")
            {

            }
            //for (const string& element : v)

            //        if (index_f == 0 && index_v != 0 && index_normal == 0)
            //        {
            //            axis.push_back(stof(element));
            //            vertices_coors[index_v] = point3((double)axis[0], (double)axis[1], (double)axis[2]);
            //        }
            //        if (index_normal != 0 && UV_map == 0)
            //        {
            //            normal_indexes.push_back(stof(element));
            //            faces_normals[index_normal] = vec3((double)normal_indexes[0], (double)normal_indexes[1], (double)normal_indexes[2]);
            //        }
            //        if (index_f != 0)
            //        {
            //            vertex_indexes.push_back(stof(element));
            //            vertices_to_faces[index_f] = vertex_indexes;
            //        }
            //    }

            //}

        Vertices_coords_size = index_v;
    }
    plik.close();

    for (int i = 0; i < Vertices_coords_size; i++) {
        cout << "Vertex: " << i + 1 << ", Coordinates: ";
        cout << vertices_coors[Vertices_coords_size][0] << ",  " << vertices_coors[Vertices_coords_size][1] << ",  " << vertices_coors[Vertices_coords_size][2];
        cout << endl;
    }

    cout << endl;
    cout << endl;
    cout << endl;

    //for (const auto& pair : faces_normals) {
    //    cout << "Face: " << pair.first << ", Normal: ";
    //    cout << pair.second << " ";
    //    Planes_to_faces[pair.first] = Plane(pair.second, vertices_coors[vertices_to_faces[pair.first][0]]);
    //    cout << endl;
    //}

    //cout << endl;
    //cout << endl;
    //cout << endl;

    //for (const auto& pair : vertices_to_faces) {
    //    cout << "Face: " << pair.first << ", Vertex list: ";
    //    for (int value : pair.second) {
    //        cout << value << " ";
    //    }
    //    cout << endl;
    //}
    //cout << endl;
    //cout << endl;
    //cout << endl;

    //for (const auto& pair : Planes_to_faces) {
    //    cout << "Face: " << pair.first << ", Plane: " << pair.second << " " << endl;
}
